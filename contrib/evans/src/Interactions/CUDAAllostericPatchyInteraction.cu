#include "hip/hip_runtime.h"
/*
 * CUDAAllostericPatchyInteraction.cu
 *
 *  Created on: 14/may/2021
 *      Author: lorenzo
 */

#include "CUDAAllostericPatchyInteraction.h"

#include "Particles/CustomParticle.h"
#include "CUDA/Lists/CUDASimpleVerletList.h"
#include "CUDA/Lists/CUDANoList.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/transform.h>

#include <hiprand/hiprand_kernel.h>

/* BEGIN CUDA */
__constant__ int MD_N[1];
__constant__ int MD_N_patch_types[1];

__constant__ int MD_N_patches[CUDAAllostericPatchyInteraction::MAX_SPECIES];
__constant__ int MD_patch_types[CUDAAllostericPatchyInteraction::MAX_SPECIES][CUDAAllostericPatchyInteraction::MAX_PATCHES];

// patch a1 values (for orientation)
__constant__ float4 MD_base_patch_a1s[CUDAAllostericPatchyInteraction::MAX_SPECIES][CUDAAllostericPatchyInteraction::MAX_PATCHES];
// TODO: consider making this texture memory? discuss with Lorenzo?

// flattened 3d array of state vars corresponding to each patch
// just a method for mapping particle.patches[p]._state_var to CUDA
__constant__ int MD_patch_var_idxs[CUDAAllostericPatchyInteraction::MAX_SPECIES][CUDAAllostericPatchyInteraction::MAX_PATCHES];

// allosteric control list
 /**
  * My notation here is, frustratingly, NOT CONSISTANT with the C++ code so here goes:
  * if indexed as MD_allosteric_controls[a][b][c]
  * a is the species that we want to get the allosteric control for
  * b is the state of the particle as an unsigned int
  * c is the patch index we're checking
  */
//__constant__ bool MD_allosteric_controls[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_STATES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
// TODO: consider making this texture memory? discuss with Lorenzo?

__constant__ float MD_sqr_rcut[1];
__constant__ float MD_sqr_rep_rcut[1];
__constant__ float MD_sqr_patch_rcut[1];
__constant__ float MD_sigma_ss[1];
__constant__ float MD_rcut_ss[1];
__constant__ float MD_lambda[1];
__constant__ float MD_A_part[1], MD_B_part[1];
__constant__ float MD_spherical_attraction_strength[1], MD_spherical_E_cut[1];

/// KF-related quantities
__constant__ bool MD_is_KF[1];
__constant__ int MD_patch_power[1];
// power delta = patch width raised to the 10th power
__constant__ float MD_patch_pow_delta[1];
__constant__ float MD_patch_pow_cosmax[1];
__constant__ float MD_patch_angular_cutoff[1];

texture<float, 1, hipReadModeElementType> tex_patchy_eps;
texture<float4, 1, hipReadModeElementType> tex_base_patches;

#include "CUDA/cuda_utils/CUDA_lr_common.cuh"

struct __align__(16) CUDA_FS_bond {
    int q;
    c_number4 force;
    c_number4 p_torque;
    c_number4 q_torque_ref_frame;
};

struct __align__(16) CUDA_FS_bond_list {
    int n_bonds;
    CUDA_FS_bond bonds[CUDAAllostericPatchyInteraction::MAX_NEIGHS];

    __device__
    CUDA_FS_bond_list() :
            n_bonds(0) {
    }
    __device__
    CUDA_FS_bond &new_bond() {
        n_bonds++;
        if(n_bonds > CUDAAllostericPatchyInteraction::MAX_NEIGHS) {
            printf("TOO MANY BONDED NEIGHBOURS, TRAGEDY\nHere is the list of neighbours:\n");
            for(int i = 0; i < n_bonds; i++) {
                printf("%d ", bonds[i].q);
            }
            printf("\n");
        }
        return bonds[n_bonds - 1];
    }
};

/**
 * Computes a two-body point interaction
 * @param ppos the position of particle p, as a quaternion. particle type can be derived from w
 * @param qpos the position of particle q, as a quaternion. particle type can be derived from w
 * @param a1 the first column of particle p rotation matrix
 * @param a2 the second column of particle p rotation matrix
 * @param a3 the third column of particle p rotation matrix
 * @param b1 the first column of particle q rotation matrix
 * @param b2 the second column of particle q rotation matrix
 * @param b3 the third column of particle q rotation matrix
 * @param F force? unclear what F.w is
 * @param torque
 * @param bonds
 * @param q_idx
 * @param box
 * @param p_activations the activation states of the patches of particle p
 * @param q_activation the activation states of the patches of particle q
 * @param p_state the binding state of particle p, where each binary digit is a patch binding state
 * @param q_state the binding state of particle q, where each binary digit is a patch binding state
 */
__device__ void _patchy_point_two_body_interaction(c_number4 &ppos,
                                                   c_number4 &qpos,
                                                   c_number4 &a1,
                                                   c_number4 &a2,
                                                   c_number4 &a3,
                                                   c_number4 &b1,
                                                   c_number4 &b2,
                                                   c_number4 &b3,
                                                   c_number4 &F,
                                                   c_number4 &torque,
                                                   CUDA_FS_bond_list *bonds,
                                                   int q_idx,
                                                   CUDABox *box,
                                                   unsigned int &p_state,
                                                   unsigned int &q_state,
                                                   bool* p_activations,
                                                   bool* q_activations) {
    int ptype = get_particle_btype(ppos);
    int qtype = get_particle_btype(qpos);

    // preliminary calcualtions - distance between the centers of the two particles
    c_number4 r = box->minimum_image(ppos, qpos);
    // get the square of the magnitude of the distance by taking the dot product of the distance with itself
    c_number sqr_r = CUDA_DOT(r, r);
    // if the distance (squared but whatever) is beyond the cutoff for two particles to interact, return
    // note that this is not the same as the patch interaction cutoff distance-square MD_sqr_patch_rcut[0]
    if(sqr_r >= MD_sqr_rcut[0]) return;

    c_number force_module = 0.f;
    c_number spherical_energy = 0.f;

    // center-center
    // if the center-center distance-squared is greater than the cutoff for repulsive force between spheres...
    // TODO: revisit - should/are DNA nanostructures be engaging in attractive intermolecular forces?
    // TODO: since they aren't single-molecules they shouldn't exhibit London Dispersion... right?
    // declare intermediate variables within blocks so they go out of scope and don't hog memory
    if(sqr_r >= MD_sqr_rep_rcut[0]) {
        // inverse of the square of the distance
        c_number ir2 = 1.f / sqr_r;
        // inverse of the 6th power of the distance - cf. lennard-jones potential
        // assume sigma = 1?
        c_number lj_part = ir2 * ir2 * ir2;
        // = -24 * LJ epsilon * (1/r^6 - 2/r^12) / r^2
        // TODO: huh? significance of the number 24?
        force_module = -24.f * MD_spherical_attraction_strength[0] * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        // Lennard-Jones potential = 4 * LJ epsilon * (1/r^12 - 1/r^6)
        spherical_energy = 4.f * MD_spherical_attraction_strength[0] * (SQR(lj_part) - lj_part);
    }
    // if the center-center distance-squared is less than the cutoff for repulsive force between spheres
    else {
        // inverse square of the distance
        c_number ir2 = 1.f / sqr_r;
        // inverse of the 6th power of the distance - cf. lennard-jones potential
        // assume sigma = 1?
        c_number lj_part = ir2 * ir2 * ir2;
        // TODO: figure out what this is
        force_module = -24.f * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        // TODO: figure out what is going on here
        //..... the 12-6 potential / epsilon plus one minus epsilon???? HUH???
        spherical_energy = 4.f * (SQR(lj_part) - lj_part) + 1.f - MD_spherical_attraction_strength[0];
    }

    // incorporate forces from sphere-sphere interaction into force
    F.x -= r.x * force_module;
    F.y -= r.y * force_module;
    F.z -= r.z * force_module;
    F.w += spherical_energy - MD_spherical_E_cut[0];

    int p_N_patches = MD_N_patches[ptype];
    int q_N_patches = MD_N_patches[qtype];

    // loop patches on particle p
    for(int p_patch = 0; p_patch < p_N_patches; p_patch++) {
        // if patch is not active, continue
        if (!p_activations[p_patch]){
//            printf("Patch %i on particle type %i cannot form binds due to patch inactive\n", p_patch, ptype);
            continue;
        }
        c_number4 p_base_patch = tex1Dfetch(tex_base_patches, p_patch + ptype * CUDAAllostericPatchyInteraction::MAX_PATCHES);

        // get position of patch p by matrix-multiplying the particle orientation and the base position
        // TODO: could move to DPS_forces and vectorize?
        c_number4 p_patch_pos = {
                a1.x * p_base_patch.x + a2.x * p_base_patch.y + a3.x * p_base_patch.z,
                a1.y * p_base_patch.x + a2.y * p_base_patch.y + a3.y * p_base_patch.z,
                a1.z * p_base_patch.x + a2.z * p_base_patch.y + a3.z * p_base_patch.z, 0.f
        };

        // loop patches on particle q
        for(int q_patch = 0; q_patch < q_N_patches; q_patch++) {
//            printf("Checking for bind between Patch %i on particle type %i & patch %i on particle type %i\n",
//                   p_patch, ptype, q_patch, qtype);
            // if patch is not active, continue
            if (!q_activations[q_patch]){
//                printf("Cannot bind to patch %i on particle type %i due to patch inactive\n", q_patch, qtype);
                continue;
            }

            c_number4 q_base_patch = tex1Dfetch(tex_base_patches, q_patch + qtype * CUDAAllostericPatchyInteraction::MAX_PATCHES);

            // get position of q patch by matrix-multiplying the particle orientation and the base position
            // TODO: move to DPS_forces and vectorize?
            c_number4 q_patch_pos = {
                    b1.x * q_base_patch.x + b2.x * q_base_patch.y + b3.x * q_base_patch.z,
                    b1.y * q_base_patch.x + b2.y * q_base_patch.y + b3.y * q_base_patch.z,
                    b1.z * q_base_patch.x + b2.z * q_base_patch.y + b3.z * q_base_patch.z, 0.f
            };

            // distance vector
            c_number4 patch_dist = {
                    r.x + q_patch_pos.x - p_patch_pos.x,
                    r.y + q_patch_pos.y - p_patch_pos.y,
                    r.z + q_patch_pos.z - p_patch_pos.z, 0.f
            };

            // get the square of the magnitude of the distance vector by dot-producting it with itself
            // TODO: it's possible that even this could be vectorized?
            c_number dist = CUDA_DOT(patch_dist, patch_dist);
//            printf("Distance: %f (compare to %f)\n", dist, MD_sqr_patch_rcut[0]);

            // if the distance-squared is greater than the square of the distance cutoff
            // (it's a 1-length array if you're curious)
            if(dist < MD_sqr_patch_rcut[0]) {

                // retrieve patch types
                int p_patch_type = MD_patch_types[ptype][p_patch];
                int q_patch_type = MD_patch_types[qtype][q_patch];


                // query the 1-d texture memory that stores the epsilon values for patch types (NOT colors!)
                c_number epsilon = tex1Dfetch(tex_patchy_eps, p_patch_type + MD_N_patch_types[0] * q_patch_type);
//                printf("Patch %i (%i) on particle type %i is within interaction distance of patch %i (%i) on particle type %i! (%f < %f, epsilon=%f)\n",
//                       p_patch, p_patch_type, ptype,
//                       q_patch, q_patch_type, qtype,
//                       dist, MD_sqr_patch_rcut[0], epsilon);
                // if the two patches can bond
                if(epsilon != (c_number) 0.f) {
                    // compute actual distance between patches
                    c_number r_p = sqrtf(dist);
                    // TODO: HUH? why isn't this redundant with the other distance conditional a few lines ago?
                    if((r_p - MD_rcut_ss[0]) < 0.f) {
//                        printf("Bond formed between patch type %i on particle type %i and patch type %i on particle type %i\n",
//                               p_patch, ptype, q_patch, qtype);

                        c_number exp_part = expf(MD_sigma_ss[0] / (r_p - MD_rcut_ss[0]));
                        c_number energy_part = epsilon * MD_A_part[0] * exp_part * (MD_B_part[0] / SQR(dist) - 1.f);

                        c_number force_mod =
                                epsilon * MD_A_part[0] * exp_part * (4.f * MD_B_part[0] / (SQR(dist) * r_p)) +
                                MD_sigma_ss[0] * energy_part / SQR(r_p - MD_rcut_ss[0]);
                        c_number4 tmp_force = patch_dist * (force_mod / r_p);

                        c_number4 p_torque = _cross(p_patch_pos, tmp_force);

                        torque -= p_torque;
                        F.x -= tmp_force.x;
                        F.y -= tmp_force.y;
                        F.z -= tmp_force.z;
                        F.w += energy_part;

                        // add bond to bonds list
                        CUDA_FS_bond &my_bond = bonds[p_patch].new_bond();

                        my_bond.q = q_idx;

                        if (r_p > MD_sigma_ss[0]) {
                            my_bond.force = tmp_force;
                            my_bond.force.w = -energy_part;
                            my_bond.p_torque = p_torque;
                            my_bond.q_torque_ref_frame = _vectors_transpose_c_number4_product(b1, b2, b3,
                                                                                              _cross(q_patch_pos,
                                                                                                     tmp_force));
                        } else {
                            my_bond.force.w = epsilon;
                        }

                        // update binding state
                        int oldState = p_state;
                        p_state = p_state | (1 << p_patch);
                        if (p_state != oldState){
//                            printf("State of particle ID %i changed from %i to %i\n", IND, oldState, p_state);
                        }
                    }
                }
            }
        }
    }
}


/**
 * Computes a two-body kern-frankel interaction
 * @param ppos the position of particle p, as a quaternion. particle type can be derived from w
 * @param qpos the position of particle q, as a quaternion. particle type can be derived from w
 * @param a1 the first column of particle p rotation matrix
 * @param a2 the second column of particle p rotation matrix
 * @param a3 the third column of particle p rotation matrix
 * @param b1 the first column of particle q rotation matrix
 * @param b2 the second column of particle q rotation matrix
 * @param b3 the third column of particle q rotation matrix
 * @param F the net force on the particle
 * @param torque
 * @param bonds
 * @param q_idx
 * @param box
 * @param p_activation the activation states of the patches of particle p
 * @param q_activations the activation states of the patches of particle q
 * @param p_state the binding state of particle p, where each binary digit is a patch binding state
 * @param q_binding_state the binding state of particle q, where each binary digit is a patch binding state
 */
__device__ void _patchy_KF_two_body_interaction(c_number4 &ppos,
                                                c_number4 &qpos,
                                                c_number4 &a1,
                                                c_number4 &a2,
                                                c_number4 &a3,
                                                c_number4 &b1,
                                                c_number4 &b2,
                                                c_number4 &b3,
                                                c_number4 &F,
                                                c_number4 &torque,
                                                CUDA_FS_bond_list *bonds,
                                                int q_idx, CUDABox *box,
                                                unsigned int &p_state,
                                                unsigned int &q_state,
                                                bool* p_activations,
                                                bool* q_activations) {
    // get particle types
    int ptype = get_particle_btype(ppos);
    int qtype = get_particle_btype(qpos);

    // r = displacement vector between positions of particles p and q
    c_number4 r = box->minimum_image(ppos, qpos);
    // sqr_r: get r^2 (square of the distance) by taking the dot product of r with itself
    c_number sqr_r = CUDA_DOT(r, r);
    if(sqr_r >= MD_sqr_rcut[0]) return;

    c_number force_module = 0.f;
    c_number spherical_energy = 0.f;

    // centre-centre
    if(sqr_r >= MD_sqr_rep_rcut[0]) {
        c_number ir2 = 1.f / sqr_r;
        c_number lj_part = ir2 * ir2 * ir2;
        force_module = -24.f * MD_spherical_attraction_strength[0] * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        spherical_energy = 4.f * MD_spherical_attraction_strength[0] * (SQR(lj_part) - lj_part);
    }
    else {
        c_number ir2 = 1.f / sqr_r;
        c_number lj_part = ir2 * ir2 * ir2;
        force_module = -24.f * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        spherical_energy = 4.f * (SQR(lj_part) - lj_part) + 1.f - MD_spherical_attraction_strength[0];
    }

    F.x -= r.x * force_module;
    F.y -= r.y * force_module;
    F.z -= r.z * force_module;
    F.w += spherical_energy - MD_spherical_E_cut[0];

    // patch-patch part
    // rmod = square-root of the distance squared = magnitude of distance between particles
    c_number rmod = sqrtf(sqr_r);
    // normalized displacement vector between particles p and q
    c_number4 r_versor = r / rmod;

    // distance between surfaces of particles p and q. sphere particle radius is fixed value 0.5
    c_number dist_surf = rmod - 1.f;
    // square of the distance between the two particle surfaces
    c_number dist_surf_sqr = SQR(dist_surf);
    //
    c_number r8b10 = SQR(SQR(dist_surf_sqr)) / MD_patch_pow_delta[0];
    c_number exp_part = -1.001f * expf(-0.5f * r8b10 * dist_surf_sqr);

    int p_N_patches = MD_N_patches[ptype];
    int q_N_patches = MD_N_patches[qtype];

    for(int p_patch = 0; p_patch < p_N_patches; p_patch++) {
        if (!p_activations[p_patch]) {
            continue;
        }
        c_number4 p_base_patch = tex1Dfetch(tex_base_patches, p_patch + ptype * CUDAAllostericPatchyInteraction::MAX_PATCHES);
        c_number4 p_patch_pos = {
                a1.x * p_base_patch.x + a2.x * p_base_patch.y + a3.x * p_base_patch.z,
                a1.y * p_base_patch.x + a2.y * p_base_patch.y + a3.y * p_base_patch.z,
                a1.z * p_base_patch.x + a2.z * p_base_patch.y + a3.z * p_base_patch.z, 0.f
        };
        p_patch_pos *= 2.f;

        // cospr = cosine of the
        c_number cospr = CUDA_DOT(p_patch_pos, r_versor);
        c_number cospr_minus_one = cospr - 1.f;
        if(cospr_minus_one < MD_patch_angular_cutoff[0]) {

            // what follows is a slightly faster way of doing (cospr - 1)^(MD_patch_power - 1) than a regular loop
            c_number part = SQR(cospr_minus_one);
            c_number cospr_base = cospr_minus_one;
            for(int i = 0; i < MD_patch_power[0] / 2 - 1; i++) {
                cospr_base *= part;
            }

            // we do this so that later we don't have to divide this number by (cospr - 1), which could be 0
            c_number cospr_part = cospr_base * cospr_minus_one;
            c_number p_mod = expf(-cospr_part / (2.f * MD_patch_pow_cosmax[0]));

            for(int q_patch = 0; q_patch < q_N_patches; q_patch++) {
                if (!q_activations[q_patch]){
                    continue;
                }
                c_number4 q_base_patch = tex1Dfetch(tex_base_patches, q_patch + qtype * CUDAAllostericPatchyInteraction::MAX_PATCHES);
                c_number4 q_patch_pos = {
                        b1.x * q_base_patch.x + b2.x * q_base_patch.y + b3.x * q_base_patch.z,
                        b1.y * q_base_patch.x + b2.y * q_base_patch.y + b3.y * q_base_patch.z,
                        b1.z * q_base_patch.x + b2.z * q_base_patch.y + b3.z * q_base_patch.z, 0.f
                };
                q_patch_pos *= 2.f;

                // cosqr
                c_number cosqr = -CUDA_DOT(q_patch_pos, r_versor);
                c_number cosqr_minus_one = cosqr - 1.f;
                if(cosqr_minus_one < MD_patch_angular_cutoff[0]) {
                    int p_patch_type = MD_patch_types[ptype][p_patch];
                    int q_patch_type = MD_patch_types[qtype][q_patch];
                    c_number epsilon = tex1Dfetch(tex_patchy_eps, p_patch_type + MD_N_patch_types[0] * q_patch_type);

                    if(epsilon != 0.f) {
                        part = SQR(cosqr_minus_one);
                        c_number cosqr_base = cosqr_minus_one;
                        for(int i = 0; i < MD_patch_power[0] / 2 - 1; i++) {
                            cosqr_base *= part;
                        }

                        c_number cosqr_part = cosqr_base * cosqr_minus_one;
                        c_number q_mod = expf(-cosqr_part / (2.f * MD_patch_pow_cosmax[0]));

                        c_number energy_part = exp_part * p_mod * q_mod;

                        // radial part
                        c_number4 radial_force = r_versor * (p_mod * q_mod * 5.f * (rmod - 1.f) * exp_part * r8b10);

                        // angular p part
                        c_number der_p = exp_part * q_mod * (0.5f * MD_patch_power[0] * p_mod * cospr_base / MD_patch_pow_cosmax[0]);
                        c_number4 p_ortho = p_patch_pos - cospr * r_versor;
                        c_number4 angular_force = p_ortho * (der_p / rmod);

                        // angular q part
                        c_number der_q = exp_part * p_mod * (-0.5f * MD_patch_power[0] * q_mod * cosqr_base / MD_patch_pow_cosmax[0]);
                        c_number4 q_ortho = q_patch_pos + cosqr * r_versor;
                        angular_force += q_ortho * (der_q / rmod);

                        c_number4 p_torque = _cross(r_versor, p_patch_pos) * der_p;
                        c_number4 q_torque = _cross(q_patch_pos, r_versor) * der_q;

                        c_number4 tot_force = radial_force + angular_force;

                        torque -= p_torque;
                        F.x -= tot_force.x;
                        F.y -= tot_force.y;
                        F.z -= tot_force.z;
                        F.w += energy_part;

                        if(energy_part < 0.f) {
                            CUDA_FS_bond &my_bond = bonds[p_patch].new_bond();

                            my_bond.force = (dist_surf < MD_sigma_ss[0]) ? angular_force : tot_force;
                            my_bond.force.w = (dist_surf < MD_sigma_ss[0]) ? epsilon * p_mod * q_mod : -energy_part;
                            my_bond.p_torque = p_torque;
                            my_bond.q_torque_ref_frame = _vectors_transpose_c_number4_product(b1, b2, b3, q_torque);
                            my_bond.q = q_idx;
                        }

                        // update particle state
                        p_state = p_state | (1 << p_patch);
                    }

                }
            }
        }
    }
}

__device__ void _three_body(CUDA_FS_bond_list *bonds, c_number4 &F, c_number4 &T, c_number4 *forces, c_number4 *torques) {
    for(int pi = 0; pi < CUDAAllostericPatchyInteraction::MAX_PATCHES; pi++) {
        CUDA_FS_bond_list &bond_list = bonds[pi];

        for(int bi = 0; bi < bond_list.n_bonds; bi++) {
            CUDA_FS_bond &b1 = bond_list.bonds[bi];
            c_number curr_energy = b1.force.w;

            for(int bj = bi + 1; bj < bond_list.n_bonds; bj++) {
                CUDA_FS_bond &b2 = bond_list.bonds[bj];
                c_number other_energy = b2.force.w;

                // the factor 2 takes into account the fact that the total pair energy is always counted twice
                F.w += 2.f * MD_lambda[0] * curr_energy * other_energy;

                // b1
                c_number factor = -MD_lambda[0] * other_energy;

                c_number4 tmp_force = b1.force * factor;
                tmp_force.w = 0.f;

                F -= tmp_force;
                LR_atomicAddXYZ(forces + b1.q, tmp_force);

                T -= factor * b1.p_torque;
                LR_atomicAddXYZ(torques + b1.q, b1.q_torque_ref_frame * factor);

                // b2
                factor = -MD_lambda[0] * curr_energy;

                tmp_force = b2.force * factor;
                tmp_force.w = 0.f;

                F -= tmp_force;
                LR_atomicAddXYZ(forces + b2.q, tmp_force);

                T -= factor * b2.p_torque;
                LR_atomicAddXYZ(torques + b2.q, b2.q_torque_ref_frame * factor);
            }
        }
    }
}

/**
 *
 * @param poss particle positions. reqd for particle type
 * @param particle_states
 * @param activations
 */
__global__ void update_patch_activations(c_number4 *poss,
                                         const unsigned int* particle_states,
                                         bool* activations_map,
                                         bool* activations){
    if(IND >= MD_N[0]) return;
    int species = get_particle_type(poss[IND]);
    for (int i = 0; i < MD_N_patches[species]; i++){
        int idx = (species * CUDAAllostericPatchyInteraction::MAX_STATES + particle_states[IND]) * CUDAAllostericPatchyInteraction::MAX_PATCHES + i;
        activations[IND * CUDAAllostericPatchyInteraction::MAX_PATCHES + i] = activations_map[idx];
    }
}

__global__ void step_particle_states(c_number4* poss,
                                     hiprandState* rand,
                                     const unsigned int* state_transition_map,
                                     unsigned int* states){
    int species = get_particle_type(poss[IND]);
    hiprandState rng = rand[IND];
    // roll on state transition table
    int table_idx = hiprand_uniform(&rng) * STATE_TRANSITION_SUBDIV;
    // big line - do state transition!
    states[IND] = state_transition_map[(species * CUDAAllostericPatchyInteraction::MAX_STATES + states[IND]) * STATE_TRANSITION_SUBDIV + table_idx];
    // lorenzo does this in the other functions
    rand[IND] = rng;
}

/** @deprecated use the version with Verlet lists
 * computes the forces for a single particle with respect to all other
 * particles in the simulation. forces + second step without lists
 * @param poss positions of all particles in the simulation
 * @param orientations orientations of all particles in the simulation
 * @param forces net forces on all particles in the simulation
 * @param three_body_forces
 * @param torques
 * @param three_body_torques
 * @param box
 * @param patch_activations
 * @param particle_states
 */
//__global__ void DPS_forces(c_number4 *poss,
//                           GPU_quat *orientations,
//                           c_number4 *forces,
//                           c_number4 *three_body_forces,
//                           c_number4 *torques,
//                           c_number4 *three_body_torques,
//                           CUDABox *box,
//                           bool* patch_activations,
//                           unsigned int *particle_states
//) {
//    if(IND >= MD_N[0]) return;
//
//    c_number4 F = forces[IND];
//    c_number4 T = torques[IND];
//    c_number4 ppos = poss[IND];
//    GPU_quat po = orientations[IND];
//    c_number4 a1, a2, a3, b1, b2, b3;
//    get_vectors_from_quat(po, a1, a2, a3);
//
//
//    // create a list of all the bonds in this iteration
//    CUDA_FS_bond_list bonds[CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
//
//    // loop through every other particle in the simulation
//    int oldState = particle_states[IND];
//    for(int j = 0; j < MD_N[0]; j++) {
//        if(j != IND) {
//            c_number4 qpos = poss[j];
//
//            GPU_quat qo = orientations[j];
//            get_vectors_from_quat(qo, b1, b2, b3);
//
//            if(MD_is_KF[0]) {
//                _patchy_KF_two_body_interaction(ppos,
//                                                qpos,
//                                                a1,
//                                                a2,
//                                                a3,
//                                                b1,
//                                                b2,
//                                                b3,
//                                                F,
//                                                T,
//                                                bonds,
//                                                j,
//                                                box,
//                                                particle_states[IND],
//                                                particle_states[j]);
//            }
//            else {
//                _patchy_point_two_body_interaction(ppos,
//                                                   qpos,
//                                                   a1,
//                                                   a2,
//                                                   a3,
//                                                   b1,
//                                                   b2,
//                                                   b3,
//                                                   F,
//                                                   T,
//                                                   bonds,
//                                                   j,
//                                                   box,
//                                                   particle_states[IND],
//                                                   particle_states[j]);
//            }
//        }
//    }
//
//    _three_body(bonds, F, T, three_body_forces, three_body_torques);
//
//    forces[IND] = F;
//    torques[IND] = _vectors_transpose_c_number4_product(a1, a2, a3, T);
////    memcpy(MD_allosteric_controls[p_type][particle_states[IND]],
////           patch_activations[IND], sizeof(bool) * )
//}

/** forces + second step with verlet lists
 * Computes the forces on particle IND
 *
 * @param poss an array of c_number4 representing the positions of all particles. index with poss[IND]
 * @param orientations an array of quaternions representing the orientations of all particles. index with orientations[IND]
 * @param forces an array of forces
 * @param three_body_forces
 * @param torques
 * @param three_body_torques
 * @param matrix_neighs
 * @param c_number_neighs
 * @param box
 * @param p_state
 * @param q_state
 */
__global__ void DPS_forces(c_number4 *poss,
                           GPU_quat *orientations,
                           c_number4 *forces,
                           c_number4 *three_body_forces,
                           c_number4 *torques,
                           c_number4 *three_body_torques,
                           int *matrix_neighs,
                           int *c_number_neighs,
                           CUDABox *box,
                           unsigned int *particle_states,
                           bool* particle_activations) {
    if(IND >= MD_N[0]) return;

    c_number4 F = forces[IND]; // copy forces value to new variable
    c_number4 T = torques[IND]; // copy torques value to new variable
    c_number4 ppos = poss[IND]; // copy positions value to new variable
    GPU_quat po = orientations[IND];
    c_number4 a1, a2, a3, b1, b2, b3;
    get_vectors_from_quat(po, a1, a2, a3);

    // create a list of bonds
    CUDA_FS_bond_list bonds[CUDAAllostericPatchyInteraction::MAX_PATCHES];

    int num_neighs = c_number_neighs[IND];
    for(int j = 0; j < num_neighs; j++) {
        int k_index = matrix_neighs[j * MD_N[0] + IND];

        c_number4 qpos = poss[k_index];

        GPU_quat qo = orientations[k_index];
        get_vectors_from_quat(qo, b1, b2, b3);

        if(MD_is_KF[0]) {
            _patchy_KF_two_body_interaction(ppos,
                                            qpos,
                                            a1,
                                            a2,
                                            a3,
                                            b1,
                                            b2,
                                            b3,
                                            F,
                                            T,
                                            bonds,
                                            k_index,
                                            box, // pass memory address
                                            particle_states[IND],
                                            particle_states[k_index],
                                            &particle_activations[IND],
                                            &particle_activations[k_index]);
        }
        else {
            _patchy_point_two_body_interaction(ppos,
                                               qpos,
                                               a1,
                                               a2,
                                               a3,
                                               b1,
                                               b2,
                                               b3,
                                               F,
                                               T,
                                               bonds,
                                               k_index,
                                               box, // pass memory address
                                               particle_states[IND],
                                               particle_states[k_index],
                                               &particle_activations[IND],
                                               &particle_activations[k_index]);
        }
    }

    _three_body(bonds, F, T, three_body_forces, three_body_torques);

    forces[IND] = F;
    torques[IND] = _vectors_transpose_c_number4_product(a1, a2, a3, T);
}

/* END CUDA PART */

#define HALF_ISQRT3 0.28867513459481292f

CUDAAllostericPatchyInteraction::CUDAAllostericPatchyInteraction() :
        CUDABaseInteraction(),
        AllostericPatchyInteraction() {
    _step = 0;
}

CUDAAllostericPatchyInteraction::~CUDAAllostericPatchyInteraction() {
    if (_d_rand_state != nullptr){
        CUDA_SAFE_CALL(hipFree(_d_rand_state));
    }
    if(_d_three_body_forces != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_three_body_forces));
    }

    if(_d_three_body_torques != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_three_body_torques));
    }

    if(_d_patchy_eps != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_patchy_eps));
        hipUnbindTexture(tex_patchy_eps);
    }

    if(_d_base_patches != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_base_patches));
        hipUnbindTexture(tex_base_patches);
    }

    if (_cu_particle_states != nullptr) {
        CUDA_SAFE_CALL(hipFree(_cu_particle_states));
    }

    if (_cu_particle_activation_map != nullptr) {
        CUDA_SAFE_CALL(hipFree(_cu_particle_activation_map));
    }

    if (_cu_state_transition_map != nullptr){
        CUDA_SAFE_CALL(hipFree(_cu_state_transition_map));
    }
}

void CUDAAllostericPatchyInteraction::get_settings(input_file &inp) {
    AllostericPatchyInteraction::get_settings(inp);

    int sort_every = 0;
    getInputInt(&inp, "CUDA_sort_every", &sort_every, 0);
}

/**
 * copies data from CPU to GPU
 */
void CUDAAllostericPatchyInteraction::sync_GPU() {
    unsigned int* binding_states = new unsigned int[cudaParticleMemoryCount()];
    // don't have to copy patch_activations bc those are derived from state
//    bool* activations = new bool[cudaParticleMemoryCount() * MAX_PATCHES];

    // loop particles
    for(int i = 0; i < realNumParticles(); i++) {
        AllostericPatchyParticle *particle = static_cast<AllostericPatchyParticle *>(CONFIG_INFO->particles()[i]);
        binding_states[i] = particle->get_state();
    }
    // copy memory to gpu
    // destination, source
//    CUDA_SAFE_CALL(hipMemcpy(_cu_particle_activation_map,
//                              activations,
//                              getActivationsArrayLength(),
//                              hipMemcpyHostToDevice));
    // copy memory to gpu
    // destination, source
    CUDA_SAFE_CALL(hipMemcpy(_cu_particle_states,
                              binding_states,
                              getBindingStateArrayLength(),
                              hipMemcpyHostToDevice));

    delete [] binding_states;
}

/**
 * copies data from GPU to CPU
 */
void CUDAAllostericPatchyInteraction::sync_host() {

    //DEBUG: do activations copy properly?
//    bool* written_activations = new bool[MAX_PATCHES * realNumParticles()];
//    CUDA_SAFE_CALL(hipMemcpy(written_activations, _cu_particle_activation_map, getActivationsArrayLength() * sizeof (bool),
//                              hipMemcpyDeviceToHost));
//    printf("Checking activations... \n");
//    for (int i = 0; i < realNumParticles(); i++){
//        AllostericPatchyParticle* pp = dynamic_cast<AllostericPatchyParticle*>(CONFIG_INFO->particles()[i]);
//        printf("Particle %i (type %i): ", i, CONFIG_INFO->particles()[i]->type);
//        for (int x = 0; x < pp->n_patches(); x++){
//            printf("%i,",written_activations[i * MAX_PATCHES + x]);
//        }
//        printf("\n");
//    }
//    delete[] written_activations;

    unsigned int* binding_states = new unsigned int[cudaParticleMemoryCount()];
    // don't have to copy patch_activations bc those are derived from state

    // copy states from gpu to cpu
    // destination, source
    CUDA_SAFE_CALL(hipMemcpy(binding_states,
                              _cu_particle_states,
                              getBindingStateArrayLength(),
                              hipMemcpyDeviceToHost));

    // loop particles
    for (int i = 0; i < realNumParticles(); i++){
        unsigned int state = binding_states[i];
        AllostericPatchyParticle* particle = static_cast<AllostericPatchyParticle*>(CONFIG_INFO->particles()[i]);
        particle->set_state(state);

        // DEBUG
//        bool* bindingState = new bool[particle->n_patches()];
//        for (int p = 0; p < particle->n_patches(); p++) {
//            bindingState[p] = particle->patches[p].bound;
//        }
//        for (int p = 0; p < particle->n_patches(); p++) {
//            bool computed_activation = particle->patch_status(bindingState, p);
//            if (computed_activation != particle->patches[p].is_active()) {
//                std::string conditional = particle->patches[p].get_allosteric_conditional();
//                throw oxDNAException("Activation state %d of particle %i, patch %i is inconsistant with allosteric control conditional %s. Binding state: %u",
//                                     particle->patches[p].is_active(),
//                                     i,
//                                     p,
//                                     conditional.c_str(),
//                                     binding_states[i]);
//            }
//        }
    }
    delete[] binding_states;
//    for(int i = 0; i < AllostericPatchySwapInteraction::_N; i++) {
//        AllostericPatchyParticle* particle = static_cast<AllostericPatchyParticle*>(CONFIG_INFO->particles()[i]);
//        short particleState;
//        CUDA_SAFE_CALL(hipMemcpy(particle_binding_states + i,
//                                  &particleState,
//                                  sizeof(short),
//                                  hipMemcpyDeviceToHost));
//
//        // loop patchesactivationState
//        for (int p = 0; p < particle->patches.size(); p++){
//            bool activationState;
//            CUDA_SAFE_CALL(hipMemcpy(activation_states + (MAX_PATCHES * i + p),
//                                      &activationState, sizeof(bool),
//                                      hipMemcpyDeviceToHost));
//
//            // the short value particleState is a binary representation of
//            // the particle state where each bit is a boolean value
//            // representing a patch binding state
//            bool newBindingState = particleState << i >= 2 << 15;
//            // set binding state
//            particle->patches[p].bound = newBindingState;
//
//            // set patch activation status
//            particle->patches[p].set_active(activationState);
//        }
//    }

}

void CUDAAllostericPatchyInteraction::cuda_init(c_number box_side, int N) {
    CUDABaseInteraction::cuda_init(box_side, N);
    AllostericPatchyInteraction::init();

    // rng (for state transitions)
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc<hiprandState>(&_d_rand_state, N * sizeof(hiprandState)));

    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_forces, N * sizeof(c_number4)));
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_torques, N * sizeof(c_number4)));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)));

    COPY_NUMBER_TO_FLOAT(MD_sqr_rcut, _sqr_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sqr_rep_rcut, _sqr_rep_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sqr_patch_rcut, _sqr_patch_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sigma_ss, _sigma_ss);
    COPY_NUMBER_TO_FLOAT(MD_rcut_ss, _rcut_ss);
    COPY_NUMBER_TO_FLOAT(MD_lambda, _lambda);
    COPY_NUMBER_TO_FLOAT(MD_A_part, _A_part);
    COPY_NUMBER_TO_FLOAT(MD_B_part, _B_part);
    COPY_NUMBER_TO_FLOAT(MD_spherical_E_cut, _spherical_E_cut);
    COPY_NUMBER_TO_FLOAT(MD_spherical_attraction_strength, _spherical_attraction_strength);

    // KF stuff
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_is_KF), &_is_KF, sizeof(bool)));

    if(_is_KF) {
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_power), &_patch_power, sizeof(int)));
        COPY_NUMBER_TO_FLOAT(MD_patch_pow_delta, _patch_pow_delta);
        COPY_NUMBER_TO_FLOAT(MD_patch_pow_cosmax, _patch_pow_cosmax);
        COPY_NUMBER_TO_FLOAT(MD_patch_angular_cutoff, _patch_angular_cutoff);
    }

    int N_strands;
    std::vector<BaseParticle *> particles(N);
    AllostericPatchyInteraction::read_topology(&N_strands, particles);

    int N_species = this->_base_particle_types.size();

    // init particle state vars
    // allocate memory for particle binding states
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_cu_particle_states,
                                           N * sizeof(unsigned int)));

    // init particle activation map
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_cu_particle_activation_map,
                                           N_species * MAX_STATES * MAX_PATCHES * sizeof(bool)));

    // init state transition map
    // to save memory, use the actual max state array length rather than the theoretical one
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_cu_state_transition_map,
                                           N_species * MAX_STATES * STATE_TRANSITION_SUBDIV * sizeof(unsigned int)));

    // malloc _patch_activations but don't need to populate it, that will happen at first step
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_patch_activations, N * MAX_PATCHES * sizeof (bool)));


    for(auto particle : particles) {
        delete particle;
    }

    if(N_species > MAX_SPECIES) {
        throw oxDNAException("PatchySwapInteraction: cannot simulate more than %d species. You can increase this number in the PatchySwapInteraction.h file", MAX_SPECIES);
    }

    uint n_patches[N_species];
    for (int i = 0; i < N_species; i++){
        n_patches[i] = _base_particle_types[i].patches.size();
    }

    // the following quantities are initialised by read_topology and hence have to be copied over to the GPU after its call
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patch_types), &_N_patch_types, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patches), n_patches, sizeof(int) * N_species));

    // patchy epsilon matrix = patch types x patch types
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_patchy_eps, _N_patch_types * _N_patch_types * sizeof(float)));
    std::vector<float> h_patchy_eps(_N_patch_types * _N_patch_types);
    // since I've deprecated the code that populates this data structure in the CPU code, do that here
    for (int i = 0; i < _N_patch_types; i++){
        for (int j = 0; j < _N_patch_types; j++){
            int idx = i * _N_patch_types + j;
            if (_base_patch_types[i].color() + _base_patch_types[j].color() == 0){
                h_patchy_eps[idx] = 1.0;
            }
            else{
                h_patchy_eps[idx] = 0;
            }
        }
    }
    CUDA_SAFE_CALL(hipMemcpy(_d_patchy_eps, h_patchy_eps.data(), h_patchy_eps.size() * sizeof(float), hipMemcpyHostToDevice));
    // bind member variable _d_patchy_eps to tex_patchy_eps
    CUDA_SAFE_CALL(hipBindTexture(NULL, tex_patchy_eps, _d_patchy_eps, h_patchy_eps.size() * sizeof(float)));

    // I mostly copied this code from this example on the nvidia website
    // https://developer.nvidia.com/blog/cuda-pro-tip-kepler-texture-objects-improve-performance-and-flexibility/

    // already malloc'd the actual space for the vars
    // create channel format descriptor for array
    // hopefully it's okay to do this locally
    // 16 bit x component, leave other components empty.

    // number of particle types x largest particle state size x num transition table subdivisions
    hipExtent extent = make_hipExtent(N_species, maxStateSize(), STATE_TRANSITION_SUBDIV);

    int N_base_patches = MAX_PATCHES * N_species;
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_base_patches, N_base_patches * sizeof(float4)));
    std::vector<float4> h_base_patches(N_base_patches, make_float4(0., 0., 0., 0.));

    int patch_var_idxs[MAX_SPECIES][MAX_PATCHES];

    // loop particle types
    for(uint ns = 0; ns < N_species; ns++) {
        AllostericPatchyParticle& particle_type = _base_particle_types[ns];
        for(uint np = 0; np < particle_type.n_patches(); np++) {
            // handle patch bp_f4 values whatever that is
            AllostericPatch& patch = particle_type.patches[np];
            float4 bp_f4 = make_float4(patch.position().x, patch.position().y, patch.position().z, 0.);
            h_base_patches[ns * MAX_PATCHES + np] = bp_f4;

            // handle patch allosteric mapping
            patch_var_idxs[ns][np] = patch.state_var();
        }
    }

    // can use N_base_patches to avoid excess copying (unused species memory space will be at end of array)
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_var_idxs), &patch_var_idxs, N_base_patches * sizeof (int), 0));

    CUDA_SAFE_CALL(hipMemcpy(_d_base_patches, h_base_patches.data(), N_base_patches * sizeof(float4), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipBindTexture(NULL, tex_base_patches, _d_base_patches, N_base_patches * sizeof(float4)));

    bool *activations = new bool[N_species * MAX_STATES * MAX_PATCHES]; // allocate temp array

    int* stateTransitionMap = new int[N_species * MAX_STATES * STATE_TRANSITION_SUBDIV];

    for(int i = 0; i < N_species; i++) {
        printf("Particle type %i\n", i);
        int n_patches = _base_particle_types[i].patches.size();

        if(n_patches > MAX_PATCHES) {
            throw oxDNAException("CUDAAllostericPatchySwapInteraction: cannot simulate particles with more than %d patches. You can increase this number in the AllostericPatchySwapInteraction.h file", MAX_PATCHES);
        }

        int patch_types[MAX_PATCHES];
        for(int p = 0; p < n_patches; p++) {
            // the patchy_epsilon matrix indexes by ID, not color!
            patch_types[p] = _base_particle_types[i].patches[p].get_id();
        }

        float4 base_patches[MAX_PATCHES];
        float4 patch_a1s[MAX_PATCHES];
        // allocate memory for patch position
        for(int p = 0; p < n_patches; p++) {
            // patch position
            LR_vector patch_position = _base_particle_types[i].patches[p].position();
            base_patches[p] = make_c_number4(patch_position.x, patch_position.y, patch_position.z, 0);
            // patch orientation
            LR_vector a1 = _base_particle_types[i].patches[p].a1();
            patch_a1s[p] = make_c_number4(a1.x, a1.y, a1.z, 0);
        }


        // fourth argument is the offset
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_types),
                                          patch_types,
                                          sizeof(int) * n_patches,
                                          i * sizeof(int) * MAX_PATCHES));
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patch_a1s),
                                          patch_a1s,
                                          sizeof(float4) * n_patches,
                                          i * sizeof(float4) * MAX_PATCHES));

        AllostericPatchyParticle& particle_type = _base_particle_types[i];

        // warning: may be nonsense if _state_transition_maps doesn't contain a key for i
        if (_state_transition_maps[i].size() > 0){
            StateTransitionMap transitionMap = _state_transition_maps[i];

            // copy activation map from base class member to the gpu
            for (int state = 0; state < particle_type.n_states(); state++) {
                // copy state transition map from base class member to the gpu
                if (_state_transition_maps.find(i) != _state_transition_maps.end()) {
                    int offset = STATE_TRANSITION_SUBDIV * MAX_STATES * i + STATE_TRANSITION_SUBDIV * state;
                    std::copy(transitionMap[state].begin(), transitionMap[state].end(), &stateTransitionMap[offset]);
                }
                memset(activations, 0, MAX_PATCHES); // wipe memory of temp array
                // do NOT use _activation_update_maps!!! that's for transitions
                for (int p = 0; p < particle_type.n_patches(); p++) {
                    int activation_var = particle_type.patches[p].activation_var();
                    int idx = i * (MAX_STATES * MAX_PATCHES) + state * MAX_PATCHES + p;
                    if (activation_var == 0) {
                        activations[idx] = true;
                    } else if (activation_var > 0) { // normal vars
                        activations[idx] = GET_BIT(state, activation_var);
                    } else { // virtual vars
                        activations[idx] = !GET_BIT(state, activation_var);
                    }
                }
            }

        }
        // copy activations array to cuda memory
        CUDA_SAFE_CALL(hipMemcpy(_cu_particle_activation_map,
                                  activations,
                                  MAX_STATES * MAX_PATCHES * sizeof(bool),
                                  hipMemcpyHostToDevice));

        // copy state transition map to cuda memory
        CUDA_SAFE_CALL(hipMemcpy(_cu_state_transition_map,
                                  stateTransitionMap,
                                  MAX_STATES * STATE_TRANSITION_SUBDIV * sizeof(unsigned int),
                                  hipMemcpyHostToDevice));

    }
    delete [] activations; // deallocate temporary array
}

/**
 *
 * @param lists list of particles
 * @param d_poss probably an array of particle positions?
 * @param d_orientations probably an array of particle orientations?
 * @param d_forces
 * @param d_torques
 * @param d_bonds
 * @param d_box
 */
void CUDAAllostericPatchyInteraction::compute_forces(CUDABaseList *lists, c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_forces, c_number4 *d_torques, LR_bonds *d_bonds, CUDABox *d_box) {
    int N = cudaParticleMemoryCount(); // number of particles
    // construct data structures for three-body computations
    thrust::device_ptr < c_number4 > t_forces = thrust::device_pointer_cast(d_forces);
    thrust::device_ptr < c_number4 > t_torques = thrust::device_pointer_cast(d_torques);
    thrust::device_ptr < c_number4 > t_three_body_forces = thrust::device_pointer_cast(_d_three_body_forces);
    thrust::device_ptr < c_number4 > t_three_body_torques = thrust::device_pointer_cast(_d_three_body_torques);
    thrust::fill_n(t_three_body_forces, N, make_c_number4(0, 0, 0, 0));
    thrust::fill_n(t_three_body_torques, N, make_c_number4(0, 0, 0, 0));

    // set patch activations
    update_patch_activations<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>(d_poss,
                                                                                    this->_cu_particle_states,
                                                                                    this->_cu_particle_activation_map,
                                                                                    this->_patch_activations);

    // DEBUG
//    printf("Beginning step %i\n", CONFIG_INFO->curr_step);
    CUDASimpleVerletList *_v_lists = dynamic_cast<CUDASimpleVerletList *>(lists);
    if(_v_lists != NULL) {
        if(_v_lists->use_edge()) throw oxDNAException("CUDAAllostericPatchySwapInteraction: use_edge is unsupported");
        else {
            DPS_forces
            <<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
                    (d_poss,
                     d_orientations,
                     d_forces,
                     _d_three_body_forces,
                     d_torques,
                     _d_three_body_torques,
                     _v_lists->d_matrix_neighs,
                     _v_lists->d_number_neighs,
                     d_box,
                     this->_cu_particle_states,
                     this->_patch_activations);
            CUT_CHECK_ERROR("DPS_forces simple_lists error");
        }
    }
    // NOTE: non-verlet version is @deprecated
    else {
//        CUDANoList *_no_lists = dynamic_cast<CUDANoList *>(lists);
//        if(_no_lists != NULL) {
//            DPS_forces
//            <<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
//                    (d_poss,
//                     d_orientations,
//                     d_forces,
//                     _d_three_body_forces,
//                     d_torques,
//                     _d_three_body_torques,
//                     d_box,
//                     this->_cu_particle_activation_map,
//                     this->_cu_particle_states);
//            CUT_CHECK_ERROR("DPS_forces no_lists error");
//        }
    }

    // add the three body contributions to the two-body forces and torques
    thrust::transform(t_forces, t_forces + N, t_three_body_forces, t_forces, thrust::plus<c_number4>());
    thrust::transform(t_torques, t_torques + N, t_three_body_torques, t_torques, thrust::plus<c_number4>());

    // do state transition step
    step_particle_states<<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>(d_poss,
                                                                                this->_d_rand_state,
                                                                                this->_cu_state_transition_map,
                                                                                this->_cu_particle_states);
}


number CUDAAllostericPatchyInteraction::pair_interaction_nonbonded(BaseParticle *p, BaseParticle *q, bool compute_r, bool update_forces) {
    if(compute_r) {
        _computed_r = _box->min_image(p->pos, q->pos);
    }

    number energy = _spherical_patchy_two_body(p, q, false, update_forces);

    if(_is_KF) {
        energy += _patchy_two_body_KF(p, q, false, update_forces);
    }
    else {
        energy += _patchy_two_body_point(p, q, false, update_forces);
    }

    return energy;
}

void CUDAAllostericPatchyInteraction::begin_energy_computation() {

}