#include "hip/hip_runtime.h"
/*
 * CUDAAllostericPatchySwapInteraction.cu
 *
 *  Created on: 15/jul/2020
 *      Author: lorenzo
 */

#include "CUDAAllostericPatchySwapInteraction.h"

#include "Particles/CustomParticle.h"
#include "CUDA/Lists/CUDASimpleVerletList.h"
#include "CUDA/Lists/CUDANoList.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/transform.h>

/* BEGIN CUDA */
__constant__ int MD_N[1];
__constant__ int MD_N_particle_types[1];

__constant__ int MD_N_patches[CUDAAllostericPatchySwapInteraction::MAX_SPECIES];
// patch positions
__constant__ float4 MD_base_patches[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
// particle epsilon values
// .... huh?
__constant__ float MD_patchy_eps[CUDAAllostericPatchySwapInteraction::MAX_SPECIES * CUDAAllostericPatchySwapInteraction::MAX_SPECIES];

// patch a1 values (for orientation)
__constant__ float4 MD_base_patch_a1s[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
// patch a2 values (for orientation)
__constant__ float4 MD_base_patch_a2s[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

// allosteric control list
/**
 * My notation here is infamously quite complecated so let's refresh:
 * if indexed as MD_allosteric_condrols[a][b][c][d],
 * a is the species that the we want to get allosteric control for
 * b is the patch on species a that we want to get allosteric control for
 * c is the current binding state of the particle that contains the patch, expressed as a binary number
 * where each digit is true if the patch at that index is bound and false otherwise
 * d is the index of the patch that is being "flipped" (bound to unbound or vice versa)
 */
__constant__ bool MD_allosteric_controls[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES][CUDAAllostericPatchySwapInteraction::MAX_STATES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

__constant__ float MD_sqr_rcut[1]; //
__constant__ float MD_sqr_rep_rcut[1]; // something something repulsive interaction
__constant__ float MD_sqr_patch_rcut[1];
__constant__ float MD_sigma_ss[1];
__constant__ float MD_rcut_ss[1];
__constant__ float MD_lambda[1];
__constant__ float MD_A_part[1], MD_B_part[1];
__constant__ float MD_spherical_attraction_strength[1], MD_spherical_E_cut[1];


#include "CUDA/cuda_utils/CUDA_lr_common.cuh"

struct __align__(16) CUDA_FS_bond {
    int q;
    bool r_p_less_than_sigma;
    c_number4 force;
    c_number4 p_torque;
    c_number4 q_torque_ref_frame;
};

struct __align__(16) CUDA_FS_bond_list {
    int n_bonds;
    CUDA_FS_bond bonds[CUDAAllostericPatchySwapInteraction::MAX_NEIGHS];

    __device__
    CUDA_FS_bond_list() :
            n_bonds(0) {
    }
    __device__
    CUDA_FS_bond &new_bond() {
        n_bonds++;
        if(n_bonds > CUDAAllostericPatchySwapInteraction::MAX_NEIGHS) {
            printf("TOO MANY BONDED NEIGHBOURS, TRAGEDY\nHere is the list of neighbours:\n");
            for(int i = 0; i < n_bonds; i++) {
                printf("%d ", bonds[i].q);
            }
            printf("\n");
        }
        return bonds[n_bonds - 1];
    }
};

/**
 * I copied this from Lorenzo's code. I have no idea what any of the parameters mean;
 * I'll add them here if I can work it out
 * @param ppos the position of particle p, represented as a number4 where w can be used to find particle species
 * @param qpos the position of particle q, represented as a number4 where w can be used to find particle species
 * @param a1 first column of the rotation matrix for particle p
 * @param a2 second column of the rotation matrix for particle p
 * @param a3 third column of the rotation matrix for particle p
 * @param b1 first column of the rotation matrix for particle q
 * @param b2 second column of the rotation matrix for particle q
 * @param b3 third column of the rotation matrix for particle q
 * @param F probably a force vector? impossible to say for sure
 * @param torque
 * @param bonds
 * @param q_idx
 * @param box
 */
__device__ void _patchy_two_body_interaction(c_number4 &ppos,
                                             c_number4 &qpos,
                                             c_number4 &a1,
                                             c_number4 &a2,
                                             c_number4 &a3,
                                             c_number4 &b1,
                                             c_number4 &b2,
                                             c_number4 &b3,
                                             c_number4 &F,
                                             c_number4 &torque,
                                             CUDA_FS_bond_list *bonds,
                                             int q_idx,
                                             CUDABox *box) {
    // derive particle types from position values
    int ptype = get_particle_btype(ppos);
    int qtype = get_particle_btype(qpos);

    // calculate... radius? from particle positions and whatever a CUDA box is
    c_number4 r = box->minimum_image(ppos, qpos);
    // square of the radius by taking the dot product of the radius vector with itself
    // (x,y,z) . (x,y,z) = x*x + y*y + z*z = x^2 + y^2 + z^2
    c_number sqr_r = CUDA_DOT(r, r);
    // if the radius is too big for particles to interact, skip
    if(sqr_r >= MD_sqr_rcut[0]) return;

    c_number force_module = 0.;
    c_number spherical_energy = 0.;

    // centre-centre
    // repulsive energy interaction from two particles occupying the same space
    // I don't need to think about this code, so I won't
    if(sqr_r >= MD_sqr_rep_rcut[0]) {
        c_number ir2 = 1.f / sqr_r;
        c_number lj_part = ir2 * ir2 * ir2;
        force_module = -24.f * MD_spherical_attraction_strength[0] * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        spherical_energy = 4.f * MD_spherical_attraction_strength[0] * (SQR(lj_part) - lj_part);
    }
    else {
        c_number ir2 = 1.f / sqr_r;
        c_number lj_part = ir2 * ir2 * ir2;
        force_module = -24.f * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        spherical_energy = 4.f * (SQR(lj_part) - lj_part) + 1 - MD_spherical_attraction_strength[0];
    }

    // subtract the repulsive force to the overall force,
    // calculating repulsive force using the radius and the force scalar thingy
    // i'm surprised there isn't a faster way to calculate this
    F.x -= r.x * force_module;
    F.y -= r.y * force_module;
    F.z -= r.z * force_module;
    F.w += spherical_energy - MD_spherical_E_cut[0];

    // retrieve number of patches on each of the two particle types
    int p_N_patches = MD_N_patches[ptype];
    int q_N_patches = MD_N_patches[qtype];

    // calcualte epsilon value for interaction between the two particle types
    // ... huh?
    c_number epsilon = MD_patchy_eps[ptype + MD_N_particle_types[0] * qtype];
    if(epsilon == (c_number) 0.f) {
        // if epsilon is 0, indicating no interaction, return
        return;
    }

    // loop patches on particle p
    for(int pi = 0; pi < p_N_patches; pi++) {
        // TODO: CHECK IF PATCH IS ACTIVE HERE
        // compute... I'm going to say patch position?
        // by taking the cross product of the base patch position and [a1 a2 a3]
        c_number4 ppatch = {
                a1.x * MD_base_patches[ptype][pi].x + a2.x * MD_base_patches[ptype][pi].y + a3.x * MD_base_patches[ptype][pi].z,
                a1.y * MD_base_patches[ptype][pi].x + a2.y * MD_base_patches[ptype][pi].y + a3.y * MD_base_patches[ptype][pi].z,
                a1.z * MD_base_patches[ptype][pi].x + a2.z * MD_base_patches[ptype][pi].y + a3.z * MD_base_patches[ptype][pi].z, 0.f
        };

        // loop patches on particle q
        for(int pj = 0; pj < q_N_patches; pj++) {
            // TODO: CHECK IF PATCH IS ACTIVE HERE
            // again, going to say patch position
            // by taking the cross product of the base patch positon and [b1 b2 b3]
            c_number4 qpatch = {
                    b1.x * MD_base_patches[qtype][pj].x + b2.x * MD_base_patches[qtype][pj].y + b3.x * MD_base_patches[qtype][pj].z,
                    b1.y * MD_base_patches[qtype][pj].x + b2.y * MD_base_patches[qtype][pj].y + b3.y * MD_base_patches[qtype][pj].z,
                    b1.z * MD_base_patches[qtype][pj].x + b2.z * MD_base_patches[qtype][pj].y + b3.z * MD_base_patches[qtype][pj].z, 0.f
            };

            // distance between the two patches
            c_number4 patch_dist = {
                    r.x + qpatch.x - ppatch.x,
                    r.y + qpatch.y - ppatch.y,
                    r.z + qpatch.z - ppatch.z, 0.f
            };

            // calculate the square of the distance between the two patches by taking
            // the dot product of the distance vector with itself
            c_number patch_dist_sqr = CUDA_DOT(patch_dist, patch_dist);
            if(patch_dist_sqr < MD_sqr_patch_rcut[0]) {
                c_number r_p = sqrtf(patch_dist_sqr);
                if((r_p - MD_rcut_ss[0]) < 0.f) {
                    c_number exp_part = expf(MD_sigma_ss[0] / (r_p - MD_rcut_ss[0]));
                    c_number energy_part = epsilon * MD_A_part[0] * exp_part * (MD_B_part[0] / SQR(patch_dist_sqr) - 1.f);

                    c_number force_mod = epsilon * MD_A_part[0] * exp_part * (4.f * MD_B_part[0] / (SQR(patch_dist_sqr) * r_p)) + MD_sigma_ss[0] * energy_part / SQR(r_p - MD_rcut_ss[0]);
                    c_number4 tmp_force = patch_dist * (force_mod / r_p);

                    // form bond; add to list
                    CUDA_FS_bond_list &bond_list = bonds[pi];
                    CUDA_FS_bond &my_bond = bond_list.new_bond();
                    // TODO: set patch activities based on state change

                    my_bond.force = tmp_force;
                    my_bond.force.w = energy_part;
                    my_bond.p_torque = _cross(ppatch, tmp_force);
                    my_bond.q_torque_ref_frame = _vectors_transpose_c_number4_product(b1, b2, b3, _cross(qpatch, tmp_force));
                    my_bond.q = q_idx;
                    my_bond.r_p_less_than_sigma = r_p < MD_sigma_ss[0];

                    torque -= my_bond.p_torque;
                    F.x -= tmp_force.x;
                    F.y -= tmp_force.y;
                    F.z -= tmp_force.z;
                    F.w += energy_part;
                }
            }
        }
    }
}

__device__ void _three_body(CUDA_FS_bond_list *bonds, c_number4 &F, c_number4 &T, c_number4 *forces, c_number4 *torques) {
    for(int pi = 0; pi < CUDAAllostericPatchySwapInteraction::MAX_PATCHES; pi++) {
        CUDA_FS_bond_list &bond_list = bonds[pi];

        for(int bi = 0; bi < bond_list.n_bonds; bi++) {
            CUDA_FS_bond &b1 = bond_list.bonds[bi];
            for(int bj = bi + 1; bj < bond_list.n_bonds; bj++) {
                CUDA_FS_bond &b2 = bond_list.bonds[bj];

                c_number curr_energy = (b1.r_p_less_than_sigma) ? 1.f : -b1.force.w;
                c_number other_energy = (b2.r_p_less_than_sigma) ? 1.f : -b2.force.w;

                // the factor 2 takes into account the fact that the pair energy is counted twice
                F.w += 2.f * MD_lambda[0] * curr_energy * other_energy;

                if(!b1.r_p_less_than_sigma) {
                    c_number factor = -MD_lambda[0] * other_energy;

                    c_number4 tmp_force = b1.force * factor;
                    tmp_force.w = 0.f;

                    F -= tmp_force;
                    LR_atomicAddXYZ(forces + b1.q, tmp_force);

                    T -= factor * b1.p_torque;
                    LR_atomicAddXYZ(torques + b1.q, b1.q_torque_ref_frame * factor);
                }

                if(!b2.r_p_less_than_sigma) {
                    c_number factor = -MD_lambda[0] * curr_energy;

                    c_number4 tmp_force = b2.force * factor;
                    tmp_force.w = 0.f;

                    F -= tmp_force;
                    LR_atomicAddXYZ(forces + b2.q, tmp_force);

                    T -= factor * b2.p_torque;
                    LR_atomicAddXYZ(torques + b2.q, b2.q_torque_ref_frame * factor);
                }
            }
        }
    }
}

/**
 * all lorenzo had to say about this is "forces + second step without lists"
 * which is actually more than he has to say about most of his functions
 * @param poss
 * @param orientations
 * @param forces
 * @param three_body_forces
 * @param torques
 * @param three_body_torques
 * @param box
 */
__global__ void PS_forces(c_number4 *poss,
                          GPU_quat *orientations,
                          c_number4 *forces,
                          c_number4 *three_body_forces,
                          c_number4 *torques,
                          c_number4 *three_body_torques,
                          CUDABox *box) {
    if(IND >= MD_N[0]) return;

    c_number4 F = forces[IND];
    c_number4 T = torques[IND];
    c_number4 ppos = poss[IND];
    GPU_quat po = orientations[IND];
    c_number4 a1, a2, a3, b1, b2, b3; // declare cols for particle orientation rotation transform matrices
    get_vectors_from_quat(po, a1, a2, a3);  // assign transform matrix cols from quaternion

    CUDA_FS_bond_list bonds[CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

    for(int j = 0; j < MD_N[0]; j++) {
        if(j != IND) {
            c_number4 qpos = poss[j];

            GPU_quat qo = orientations[j];
            get_vectors_from_quat(qo, b1, b2, b3);
            _patchy_two_body_interaction(ppos, qpos, a1, a2, a3, b1, b2, b3, F, T, bonds, j, box);
        }
    }

    _three_body(bonds, F, T, three_body_forces, three_body_torques);

    forces[IND] = F;
    torques[IND] = _vectors_transpose_c_number4_product(a1, a2, a3, T);
}
/**
 * lorenzo speaketh: "forces + second step with verlet lists"
 * @param poss
 * @param orientations
 * @param forces
 * @param three_body_forces
 * @param torques
 * @param three_body_torques
 * @param matrix_neighs
 * @param c_number_neighs
 * @param box
 */
__global__ void PS_forces(c_number4 *poss,
                          GPU_quat *orientations,
                          c_number4 *forces,
                          c_number4 *three_body_forces,
                          c_number4 *torques,
                          c_number4 *three_body_torques,
                          int *matrix_neighs,
                          int *c_number_neighs,
                          CUDABox *box) {
    if(IND >= MD_N[0]) return;

    c_number4 F = forces[IND];
    c_number4 T = torques[IND];
    c_number4 ppos = poss[IND];
    GPU_quat po = orientations[IND];
    c_number4 a1, a2, a3, b1, b2, b3; // declare cols for particle orientation rotation transform matrices
    get_vectors_from_quat(po, a1, a2, a3);  // assign transform matrix cols from quaternion

    CUDA_FS_bond_list bonds[CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

    int num_neighs = c_number_neighs[IND];
    for(int j = 0; j < num_neighs; j++) {
        int k_index = matrix_neighs[j * MD_N[0] + IND];

        c_number4 qpos = poss[k_index];

        GPU_quat qo = orientations[k_index];
        get_vectors_from_quat(qo, b1, b2, b3);
        _patchy_two_body_interaction(ppos, qpos, a1, a2, a3, b1, b2, b3, F, T, bonds, k_index, box);
    }

    _three_body(bonds, F, T, three_body_forces, three_body_torques);

    forces[IND] = F;
    torques[IND] = _vectors_transpose_c_number4_product(a1, a2, a3, T);
}

/* END CUDA PART */

#define HALF_ISQRT3 0.28867513459481292f

CUDAAllostericPatchySwapInteraction::CUDAAllostericPatchySwapInteraction() :
        CUDABaseInteraction(),
        AllostericPatchySwapInteraction() {
    _d_three_body_forces = _d_three_body_torques = NULL;
    _step = 0;
}

CUDAAllostericPatchySwapInteraction::~CUDAAllostericPatchySwapInteraction() {
    if(_d_three_body_forces != NULL) {
        CUDA_SAFE_CALL(hipFree(_d_three_body_forces));
    }
    if(_d_three_body_torques != NULL) {
        CUDA_SAFE_CALL(hipFree(_d_three_body_torques));
    }
}

void CUDAAllostericPatchySwapInteraction::get_settings(input_file &inp) {
    AllostericPatchySwapInteraction::get_settings(inp);

    int sort_every = 0;
    getInputInt(&inp, "CUDA_sort_every", &sort_every, 0);
}

void CUDAAllostericPatchySwapInteraction::cuda_init(c_number box_side, int N) {
    CUDABaseInteraction::cuda_init(box_side, N);
    AllostericPatchySwapInteraction::init();

    // throw an error if there are too many species
    if(_N_particle_types > MAX_SPECIES) {
        throw oxDNAException("PatchySwapInteraction: cannot simulate more than %d species. You can increase this number in the PatchySwapInteraction.h file", MAX_SPECIES);
    }

    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_forces, N * sizeof(c_number4)));
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_torques, N * sizeof(c_number4)));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)));

    COPY_NUMBER_TO_FLOAT(MD_sqr_rcut, _sqr_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sqr_rep_rcut, _sqr_rep_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sqr_patch_rcut, _sqr_patch_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sigma_ss, _sigma_ss);
    COPY_NUMBER_TO_FLOAT(MD_rcut_ss, _rcut_ss);
    COPY_NUMBER_TO_FLOAT(MD_lambda, _lambda);
    COPY_NUMBER_TO_FLOAT(MD_A_part, _A_part);
    COPY_NUMBER_TO_FLOAT(MD_B_part, _B_part);
    COPY_NUMBER_TO_FLOAT(MD_spherical_E_cut, _spherical_E_cut);
    COPY_NUMBER_TO_FLOAT(MD_spherical_attraction_strength, _spherical_attraction_strength);

    int N_strands;
    std::vector<BaseParticle *> particles(N);
    AllostericPatchySwapInteraction::read_topology(&N_strands, particles);
    for(auto particle : particles) {
        delete particle;
    }

    // the following quantities are initialised by read_topology and hence have to be copied over to the GPU after its call
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N_particle_types), &_N_particle_types, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patches), _N_patches.data(), sizeof(int) * _N_patches.size()));
    COPY_ARRAY_TO_CONSTANT(MD_patchy_eps, _patchy_eps.data(), _patchy_eps.size());

    // for each particle type
    for(int i = 0; i < _N_particle_types; i++) {
        int n_patches = _base_particle_types[i].n_patches();

        // throw an error if we've exceeded the maximum number of patches
        if(n_patches > MAX_PATCHES) {
            throw oxDNAException("PatchySwapInteraction: cannot simulate particles with more than %d patches. You can increase this number in the PatchySwapInteraction.h file", MAX_PATCHES);
        }

        float4 base_patches[MAX_PATCHES];
        float4 patch_a1s[MAX_PATCHES];
        float4 patch_a2s[MAX_PATCHES];
        // allocate memory for patch position
        for(int p = 0; p < n_patches; p++) {
            // patch position
            LR_vector patch_position = _base_particle_types[i].patches[p].position();
            base_patches[p] = make_c_number4(patch_position.x, patch_position.y, patch_position.z, 0);
            // patch orientation
            LR_vector a1 = _base_particle_types[i].patches[p].a1();
            LR_vector a2 = _base_particle_types[i].patches[i].a2();
            patch_a1s[p] = make_c_number4(a1.x, a1.y, a1.z, 0);
            patch_a2s[p] = make_c_number4(a2.x, a2.y, a2.z, 0);

            // time to deal with allostery!
            bool patches_allosteric_flips[MAX_STATES][MAX_PATCHES];

            bool state[MAX_PATCHES];
            for (int q = 0; q < MAX_STATES; q++){
                // each unique state can be expressed as an MAX_STATES-digit binary number where
                // each digit is a patch binding state

                // first decode state
                int n = q;
                for (int x = 0; x < MAX_PATCHES; x++){
                    state[x] = n & 1;
                    n /= 2;
                }

                // encode flip value for each patch x in relation to q
                for (int x = 0; x < MAX_PATCHES; x++) {
                    // get the particle state change originating at `state` when patch `x` is flipped
                    ParticleStateChange state_change(state, MAX_PATCHES, x);
                    // get the state change result, specifically the effect on patch p
                    patches_allosteric_flips[q][x] = _base_particle_types[i].get_state_change_result(state_change)[p];
                }
            }
            // I'm like 79% sure these values are right
            int allo_mem_count = sizeof(bool) * MAX_STATES * MAX_PATCHES;
            int allo_mem_offset = (i * MAX_PATCHES + p) * allo_mem_offset;
            CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_allosteric_controls), patches_allosteric_flips, allo_mem_count, allo_mem_offset));
        }

        // fourth argument is the offset
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patches), base_patches, sizeof(float4) * n_patches, i * sizeof(float4) * MAX_PATCHES));
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patch_a1s), patch_a1s, sizeof(float4) * n_patches, i * sizeof(float4) * MAX_PATCHES));
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patch_a2s), patch_a2s, sizeof(float4) * n_patches, i * sizeof(float4) * MAX_PATCHES));
    }
}
/**
 * Function called from CUDA thing that computes forces
 * What do the parameters mean? Some mysteries may never be solved
 * @param lists a pointer to the head of an array of lists?
 * @param d_poss pointer to the head of an array of particle positions?
 * @param d_orientations pointer to the head of an array of particle orientations?
 * @param d_forces
 * @param d_torques
 * @param d_bonds
 * @param d_box
 */
void CUDAAllostericPatchySwapInteraction::compute_forces(CUDABaseList *lists, c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_forces, c_number4 *d_torques, LR_bonds *d_bonds, CUDABox *d_box) {
    int N = CUDABaseInteraction::_N;
    thrust::device_ptr < c_number4 > t_forces = thrust::device_pointer_cast(d_forces);
    thrust::device_ptr < c_number4 > t_torques = thrust::device_pointer_cast(d_torques);
    thrust::device_ptr < c_number4 > t_three_body_forces = thrust::device_pointer_cast(_d_three_body_forces);
    thrust::device_ptr < c_number4 > t_three_body_torques = thrust::device_pointer_cast(_d_three_body_torques);
    thrust::fill_n(t_three_body_forces, N, make_c_number4(0, 0, 0, 0));
    thrust::fill_n(t_three_body_torques, N, make_c_number4(0, 0, 0, 0));

    // This is some pchem nonsense
    CUDASimpleVerletList *_v_lists = dynamic_cast<CUDASimpleVerletList *>(lists);
    if(_v_lists != NULL) {
        if(_v_lists->use_edge()) throw oxDNAException("CUDAAllostericPatchySwapInteraction: use_edge is unsupported");
        else {
            PS_forces
            <<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
                    (d_poss, d_orientations, d_forces, _d_three_body_forces,  d_torques, _d_three_body_torques, _v_lists->d_matrix_neighs, _v_lists->d_number_neighs, d_box);
            CUT_CHECK_ERROR("PS_forces simple_lists error");
        }
    }
    else {
        CUDANoList *_no_lists = dynamic_cast<CUDANoList *>(lists);
        if(_no_lists != NULL) {
            PS_forces
            <<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
                    (d_poss, d_orientations, d_forces, _d_three_body_forces,  d_torques, _d_three_body_torques, d_box);
            CUT_CHECK_ERROR("PS_forces no_lists error");
        }
    }

    // add the three body contributions to the two-body forces and torques
    thrust::transform(t_forces, t_forces + N, t_three_body_forces, t_forces, thrust::plus<c_number4>());
    thrust::transform(t_torques, t_torques + N, t_three_body_torques, t_torques, thrust::plus<c_number4>());
}
