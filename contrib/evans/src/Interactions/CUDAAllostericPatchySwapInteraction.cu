#include "hip/hip_runtime.h"
/*
 * CUDAAllostericPatchySwapInteraction.cu
 *
 *  Created on: 14/may/2021
 *      Author: lorenzo
 */

#include "CUDAAllostericPatchySwapInteraction.h"

#include "Particles/CustomParticle.h"
#include "CUDA/Lists/CUDASimpleVerletList.h"
#include "CUDA/Lists/CUDANoList.h"

#include <thrust/device_ptr.h>
#include <thrust/fill.h>
#include <thrust/transform.h>

/* BEGIN CUDA */
__constant__ int MD_N[1];
__constant__ int MD_N_patch_types[1];

__constant__ int MD_N_patches[CUDAAllostericPatchySwapInteraction::MAX_SPECIES];
__constant__ int MD_patch_types[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

// patch a1 values (for orientation)
__constant__ float4 MD_base_patch_a1s[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
// patch a2 values (for orientation)
__constant__ float4 MD_base_patch_a2s[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
// TODO: consider making this texture memory? discuss with Lorenzo?

// allosteric control list
 /**
  * My notation here is, frustratingly, NOT CONSISTANT with the C++ code so here goes:
  * if indexed as MD_allosteric_controls[a][b][c]
  * a is the species that we want to get the allosteric control for
  * b is the state of the particle as an unsigned int
  * c is the patch index we're checking
  */
__constant__ bool MD_allosteric_controls[CUDAAllostericPatchySwapInteraction::MAX_SPECIES][CUDAAllostericPatchySwapInteraction::MAX_STATES][CUDAAllostericPatchySwapInteraction::MAX_PATCHES];
// TODO: consider making this texture memory? discuss with Lorenzo?

__constant__ float MD_sqr_rcut[1];
__constant__ float MD_sqr_rep_rcut[1];
__constant__ float MD_sqr_patch_rcut[1];
__constant__ float MD_sigma_ss[1];
__constant__ float MD_rcut_ss[1];
__constant__ float MD_lambda[1];
__constant__ float MD_A_part[1], MD_B_part[1];
__constant__ float MD_spherical_attraction_strength[1], MD_spherical_E_cut[1];

/// KF-related quantities
__constant__ bool MD_is_KF[1];
__constant__ int MD_patch_power[1];
__constant__ float MD_patch_pow_delta[1];
__constant__ float MD_patch_pow_cosmax[1];
__constant__ float MD_patch_angular_cutoff[1];

texture<float, 1, hipReadModeElementType> tex_patchy_eps;
texture<float4, 1, hipReadModeElementType> tex_base_patches;

#include "CUDA/cuda_utils/CUDA_lr_common.cuh"

struct __align__(16) CUDA_FS_bond {
    int q;
    c_number4 force;
    c_number4 p_torque;
    c_number4 q_torque_ref_frame;
};

struct __align__(16) CUDA_FS_bond_list {
    int n_bonds;
    CUDA_FS_bond bonds[CUDAAllostericPatchySwapInteraction::MAX_NEIGHS];

    __device__
    CUDA_FS_bond_list() :
            n_bonds(0) {
    }
    __device__
    CUDA_FS_bond &new_bond() {
        n_bonds++;
        if(n_bonds > CUDAAllostericPatchySwapInteraction::MAX_NEIGHS) {
            printf("TOO MANY BONDED NEIGHBOURS, TRAGEDY\nHere is the list of neighbours:\n");
            for(int i = 0; i < n_bonds; i++) {
                printf("%d ", bonds[i].q);
            }
            printf("\n");
        }
        return bonds[n_bonds - 1];
    }
};

/**
 * Computes a two-body point interaction
 * @param ppos the position of particle p, as a quaternion. particle type can be derived from w
 * @param qpos the position of particle q, as a quaternion. particle type can be derived from w
 * @param a1 the first column of particle p rotation matrix
 * @param a2 the second column of particle p rotation matrix
 * @param a3 the third column of particle p rotation matrix
 * @param b1 the first column of particle q rotation matrix
 * @param b2 the second column of particle q rotation matrix
 * @param b3 the third column of particle q rotation matrix
 * @param F force? unclear what F.w is
 * @param torque
 * @param bonds
 * @param q_idx
 * @param box
 * @param p_activations the activation states of the patches of particle p
 * @param q_activation the activation states of the patches of particle q
 * @param p_state the binding state of particle p, where each binary digit is a patch binding state
 * @param q_state the binding state of particle q, where each binary digit is a patch binding state
 */
__device__ void
_patchy_point_two_body_interaction(c_number4 &ppos,
                                   c_number4 &qpos,
                                   c_number4 &a1,
                                   c_number4 &a2,
                                   c_number4 &a3,
                                   c_number4 &b1,
                                   c_number4 &b2,
                                   c_number4 &b3,
                                   c_number4 &F,
                                   c_number4 &torque,
                                   CUDA_FS_bond_list *bonds,
                                   int q_idx,
                                   CUDABox *box,
                                   const bool *p_activations,
                                   const bool *q_activation,
                                   unsigned int &p_state) {
    int ptype = get_particle_btype(ppos);
    int qtype = get_particle_btype(qpos);

    // preliminary calcualtions - distance between the centers of the two particles
    c_number4 r = box->minimum_image(ppos, qpos);
    // get the square of the magnitude of the distance by taking the dot product of the distance with itself
    c_number sqr_r = CUDA_DOT(r, r);
    // if the distance (squared but whatever) is beyond the cutoff for two particles to interact, return
    // note that this is not the same as the patch interaction cutoff distance-square MD_sqr_patch_rcut[0]
    if(sqr_r >= MD_sqr_rcut[0]) return;

    c_number force_module = 0.f;
    c_number spherical_energy = 0.f;

    // center-center
    // if the center-center distance-squared is greater than the cutoff for repulsive force between spheres...
    // TODO: revisit - should/are DNA nanostructures be engaging in attractive intermolecular forces?
    // TODO: since they aren't single-molecules they shouldn't exhibit London Dispersion... right?
    // declare intermediate variables within blocks so they go out of scope and don't hog memory
    if(sqr_r >= MD_sqr_rep_rcut[0]) {
        // inverse of the square of the distance
        c_number ir2 = 1.f / sqr_r;
        // inverse of the 6th power of the distance - cf. lennard-jones potential
        // assume sigma = 1?
        c_number lj_part = ir2 * ir2 * ir2;
        // = -24 * LJ epsilon * (1/r^6 - 2/r^12) / r^2
        // TODO: huh? significance of the number 24?
        force_module = -24.f * MD_spherical_attraction_strength[0] * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        // Lennard-Jones potential = 4 * LJ epsilon * (1/r^12 - 1/r^6)
        spherical_energy = 4.f * MD_spherical_attraction_strength[0] * (SQR(lj_part) - lj_part);
    }
    // if the center-center distance-squared is less than the cutoff for repulsive force between spheres
    else {
        // inverse square of the distance
        c_number ir2 = 1.f / sqr_r;
        // inverse of the 6th power of the distance - cf. lennard-jones potential
        // assume sigma = 1?
        c_number lj_part = ir2 * ir2 * ir2;
        // TODO: figure out what this is
        force_module = -24.f * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        // TODO: figure out what is going on here
        //..... the 12-6 potential / epsilon plus one minus epsilon???? HUH???
        spherical_energy = 4.f * (SQR(lj_part) - lj_part) + 1.f - MD_spherical_attraction_strength[0];
    }

    // incorporate forces from sphere-sphere interaction into force
    F.x -= r.x * force_module;
    F.y -= r.y * force_module;
    F.z -= r.z * force_module;
    F.w += spherical_energy - MD_spherical_E_cut[0];

    int p_N_patches = MD_N_patches[ptype];
    int q_N_patches = MD_N_patches[qtype];

    // loop patches on particle p
    for(int p_patch = 0; p_patch < p_N_patches; p_patch++) {
        // if patch is not active, continue
        if (!p_activations[p_patch]){
//            printf("Patch %i on particle type %i cannot form binds due to patch inactive\n", p_patch, ptype);
            continue;
        }
        c_number4 p_base_patch = tex1Dfetch(tex_base_patches, p_patch + ptype * CUDAAllostericPatchySwapInteraction::MAX_PATCHES);

        // get position of patch p by matrix-multiplying the particle orientation and the base position
        // TODO: could move to DPS_forces and vectorize?
        c_number4 p_patch_pos = {
                a1.x * p_base_patch.x + a2.x * p_base_patch.y + a3.x * p_base_patch.z,
                a1.y * p_base_patch.x + a2.y * p_base_patch.y + a3.y * p_base_patch.z,
                a1.z * p_base_patch.x + a2.z * p_base_patch.y + a3.z * p_base_patch.z, 0.f
        };

        // loop patches on particle q
        for(int q_patch = 0; q_patch < q_N_patches; q_patch++) {
//            printf("Checking for bind between Patch %i on particle type %i & patch %i on particle type %i\n",
//                   p_patch, ptype, q_patch, qtype);
            // if patch is not active, continue
            if (!q_activation[q_patch]){
//                printf("Cannot bind to patch %i on particle type %i due to patch inactive\n", q_patch, qtype);
                continue;
            }

            c_number4 q_base_patch = tex1Dfetch(tex_base_patches, q_patch + qtype * CUDAAllostericPatchySwapInteraction::MAX_PATCHES);

            // get position of q patch by matrix-multiplying the particle orientation and the base position
            // TODO: move to DPS_forces and vectorize?
            c_number4 q_patch_pos = {
                    b1.x * q_base_patch.x + b2.x * q_base_patch.y + b3.x * q_base_patch.z,
                    b1.y * q_base_patch.x + b2.y * q_base_patch.y + b3.y * q_base_patch.z,
                    b1.z * q_base_patch.x + b2.z * q_base_patch.y + b3.z * q_base_patch.z, 0.f
            };

            // distance vector
            c_number4 patch_dist = {
                    r.x + q_patch_pos.x - p_patch_pos.x,
                    r.y + q_patch_pos.y - p_patch_pos.y,
                    r.z + q_patch_pos.z - p_patch_pos.z, 0.f
            };

            // get the square of the magnitude of the distance vector by dot-producting it with itself
            // TODO: it's possible that even this could be vectorized?
            c_number dist = CUDA_DOT(patch_dist, patch_dist);
//            printf("Distance: %f (compare to %f)\n", dist, MD_sqr_patch_rcut[0]);

            // if the distance-squared is greater than the square of the distance cutoff
            // (it's a 1-length array if you're curious)
            if(dist < MD_sqr_patch_rcut[0]) {

                // retrieve patch types
                int p_patch_type = MD_patch_types[ptype][p_patch];
                int q_patch_type = MD_patch_types[qtype][q_patch];

                // query the 1-d texture memory that stores the epsilon values for patch types (NOT colors!)
                c_number epsilon = tex1Dfetch(tex_patchy_eps, p_patch_type + MD_N_patch_types[0] * q_patch_type);
//                printf("Patch %i (%i) on particle type %i is within interaction distance of patch %i (%i) on particle type %i! (%f < %f, epsilon=%f)\n",
//                       p_patch, p_patch_type, ptype,
//                       q_patch, q_patch_type, qtype,
//                       dist, MD_sqr_patch_rcut[0], epsilon);
                // if the two patches can bond
                if(epsilon != (c_number) 0.f) {
                    // compute actual distance between patches
                    c_number r_p = sqrtf(dist);
                    // TODO: HUH? why isn't this redundant with the other distance conditional a few lines ago?
                    if((r_p - MD_rcut_ss[0]) < 0.f) {
//                        printf("Bond formed between patch type %i on particle type %i and patch type %i on particle type %i\n",
//                               p_patch, ptype, q_patch, qtype);

                        c_number exp_part = expf(MD_sigma_ss[0] / (r_p - MD_rcut_ss[0]));
                        c_number energy_part = epsilon * MD_A_part[0] * exp_part * (MD_B_part[0] / SQR(dist) - 1.f);

                        c_number force_mod =
                                epsilon * MD_A_part[0] * exp_part * (4.f * MD_B_part[0] / (SQR(dist) * r_p)) +
                                MD_sigma_ss[0] * energy_part / SQR(r_p - MD_rcut_ss[0]);
                        c_number4 tmp_force = patch_dist * (force_mod / r_p);

                        c_number4 p_torque = _cross(p_patch_pos, tmp_force);

                        torque -= p_torque;
                        F.x -= tmp_force.x;
                        F.y -= tmp_force.y;
                        F.z -= tmp_force.z;
                        F.w += energy_part;

                        // add bond to bonds list
                        CUDA_FS_bond &my_bond = bonds[p_patch].new_bond();

                        my_bond.q = q_idx;

                        if (r_p > MD_sigma_ss[0]) {
                            my_bond.force = tmp_force;
                            my_bond.force.w = -energy_part;
                            my_bond.p_torque = p_torque;
                            my_bond.q_torque_ref_frame = _vectors_transpose_c_number4_product(b1, b2, b3,
                                                                                              _cross(q_patch_pos,
                                                                                                     tmp_force));
                        } else {
                            my_bond.force.w = epsilon;
                        }

                        // update binding state
                        p_state = p_state | (1 << p_patch);
                    }
                }
            }
        }
    }
}


/**
 * Computes a two-body kern-frankel interaction
 * @param ppos the position of particle p, as a quaternion. particle type can be derived from w
 * @param qpos the position of particle q, as a quaternion. particle type can be derived from w
 * @param a1 the first column of particle p rotation matrix
 * @param a2 the second column of particle p rotation matrix
 * @param a3 the third column of particle p rotation matrix
 * @param b1 the first column of particle q rotation matrix
 * @param b2 the second column of particle q rotation matrix
 * @param b3 the third column of particle q rotation matrix
 * @param F the net force on the particle
 * @param torque
 * @param bonds
 * @param q_idx
 * @param box
 * @param p_activation the activation states of the patches of particle p
 * @param q_activations the activation states of the patches of particle q
 * @param p_binding_state the binding state of particle p, where each binary digit is a patch binding state
 * @param q_binding_state the binding state of particle q, where each binary digit is a patch binding state
 */
__device__ void
_patchy_KF_two_body_interaction(c_number4 &ppos,
                                c_number4 &qpos,
                                c_number4 &a1,
                                c_number4 &a2,
                                c_number4 &a3,
                                c_number4 &b1,
                                c_number4 &b2,
                                c_number4 &b3,
                                c_number4 &F,
                                c_number4 &torque,
                                CUDA_FS_bond_list *bonds, int q_idx, CUDABox *box, const bool *p_activation,
                                const bool *q_activations, unsigned int &p_binding_state) {
    int ptype = get_particle_btype(ppos);
    int qtype = get_particle_btype(qpos);

    c_number4 r = box->minimum_image(ppos, qpos);
    c_number sqr_r = CUDA_DOT(r, r);
    if(sqr_r >= MD_sqr_rcut[0]) return;

    c_number force_module = 0.f;
    c_number spherical_energy = 0.f;

    // centre-centre
    if(sqr_r >= MD_sqr_rep_rcut[0]) {
        c_number ir2 = 1.f / sqr_r;
        c_number lj_part = ir2 * ir2 * ir2;
        force_module = -24.f * MD_spherical_attraction_strength[0] * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        spherical_energy = 4.f * MD_spherical_attraction_strength[0] * (SQR(lj_part) - lj_part);
    }
    else {
        c_number ir2 = 1.f / sqr_r;
        c_number lj_part = ir2 * ir2 * ir2;
        force_module = -24.f * (lj_part - 2.f * SQR(lj_part)) / sqr_r;
        spherical_energy = 4.f * (SQR(lj_part) - lj_part) + 1.f - MD_spherical_attraction_strength[0];
    }

    F.x -= r.x * force_module;
    F.y -= r.y * force_module;
    F.z -= r.z * force_module;
    F.w += spherical_energy - MD_spherical_E_cut[0];

    // patch-patch part
    c_number rmod = sqrtf(sqr_r);
    c_number4 r_versor = r / rmod;

    c_number dist_surf = rmod - 1.f;
    c_number dist_surf_sqr = SQR(dist_surf);
    c_number r8b10 = SQR(SQR(dist_surf_sqr)) / MD_patch_pow_delta[0];
    c_number exp_part = -1.001f * expf(-0.5f * r8b10 * dist_surf_sqr);

    int p_N_patches = MD_N_patches[ptype];
    int q_N_patches = MD_N_patches[qtype];

    for(int p_patch = 0; p_patch < p_N_patches; p_patch++) {
        if (!p_activation[p_patch]){
            continue;
        }
        c_number4 p_base_patch = tex1Dfetch(tex_base_patches, p_patch + ptype * CUDAAllostericPatchySwapInteraction::MAX_PATCHES);
        c_number4 p_patch_pos = {
                a1.x * p_base_patch.x + a2.x * p_base_patch.y + a3.x * p_base_patch.z,
                a1.y * p_base_patch.x + a2.y * p_base_patch.y + a3.y * p_base_patch.z,
                a1.z * p_base_patch.x + a2.z * p_base_patch.y + a3.z * p_base_patch.z, 0.f
        };
        p_patch_pos *= 2.f;

        c_number cospr = CUDA_DOT(p_patch_pos, r_versor);
        c_number cospr_minus_one = cospr - 1.f;
        if(cospr_minus_one < MD_patch_angular_cutoff[0]) {

            // what follows is a slightly faster way of doing (cospr - 1)^(MD_patch_power - 1) than a regular loop
            c_number part = SQR(cospr_minus_one);
            c_number cospr_base = cospr_minus_one;
            for(int i = 0; i < MD_patch_power[0] / 2 - 1; i++) {
                cospr_base *= part;
            }

            // we do this so that later we don't have to divide this number by (cospr - 1), which could be 0
            c_number cospr_part = cospr_base * cospr_minus_one;
            c_number p_mod = expf(-cospr_part / (2.f * MD_patch_pow_cosmax[0]));

            for(int q_patch = 0; q_patch < q_N_patches; q_patch++) {
                if (!q_activations[q_patch]){
                    continue;
                }
                c_number4 q_base_patch = tex1Dfetch(tex_base_patches, q_patch + qtype * CUDAAllostericPatchySwapInteraction::MAX_PATCHES);
                c_number4 q_patch_pos = {
                        b1.x * q_base_patch.x + b2.x * q_base_patch.y + b3.x * q_base_patch.z,
                        b1.y * q_base_patch.x + b2.y * q_base_patch.y + b3.y * q_base_patch.z,
                        b1.z * q_base_patch.x + b2.z * q_base_patch.y + b3.z * q_base_patch.z, 0.f
                };
                q_patch_pos *= 2.f;

                c_number cosqr = -CUDA_DOT(q_patch_pos, r_versor);
                c_number cosqr_minus_one = cosqr - 1.f;
                if(cosqr_minus_one < MD_patch_angular_cutoff[0]) {
                    int p_patch_type = MD_patch_types[ptype][p_patch];
                    int q_patch_type = MD_patch_types[qtype][q_patch];
                    c_number epsilon = tex1Dfetch(tex_patchy_eps, p_patch_type + MD_N_patch_types[0] * q_patch_type);

                    if(epsilon != 0.f) {
                        part = SQR(cosqr_minus_one);
                        c_number cosqr_base = cosqr_minus_one;
                        for(int i = 0; i < MD_patch_power[0] / 2 - 1; i++) {
                            cosqr_base *= part;
                        }

                        c_number cosqr_part = cosqr_base * cosqr_minus_one;
                        c_number q_mod = expf(-cosqr_part / (2.f * MD_patch_pow_cosmax[0]));

                        c_number energy_part = exp_part * p_mod * q_mod;

                        // radial part
                        c_number4 radial_force = r_versor * (p_mod * q_mod * 5.f * (rmod - 1.f) * exp_part * r8b10);

                        // angular p part
                        c_number der_p = exp_part * q_mod * (0.5f * MD_patch_power[0] * p_mod * cospr_base / MD_patch_pow_cosmax[0]);
                        c_number4 p_ortho = p_patch_pos - cospr * r_versor;
                        c_number4 angular_force = p_ortho * (der_p / rmod);

                        // angular q part
                        c_number der_q = exp_part * p_mod * (-0.5f * MD_patch_power[0] * q_mod * cosqr_base / MD_patch_pow_cosmax[0]);
                        c_number4 q_ortho = q_patch_pos + cosqr * r_versor;
                        angular_force += q_ortho * (der_q / rmod);

                        c_number4 p_torque = _cross(r_versor, p_patch_pos) * der_p;
                        c_number4 q_torque = _cross(q_patch_pos, r_versor) * der_q;

                        c_number4 tot_force = radial_force + angular_force;

                        torque -= p_torque;
                        F.x -= tot_force.x;
                        F.y -= tot_force.y;
                        F.z -= tot_force.z;
                        F.w += energy_part;

                        if(energy_part < 0.f) {
                            CUDA_FS_bond &my_bond = bonds[p_patch].new_bond();

                            my_bond.force = (dist_surf < MD_sigma_ss[0]) ? angular_force : tot_force;
                            my_bond.force.w = (dist_surf < MD_sigma_ss[0]) ? epsilon * p_mod * q_mod : -energy_part;
                            my_bond.p_torque = p_torque;
                            my_bond.q_torque_ref_frame = _vectors_transpose_c_number4_product(b1, b2, b3, q_torque);
                            my_bond.q = q_idx;
                        }

                        // update particle state
                        p_binding_state = p_binding_state | (1 << p_patch);
                    }

                }
            }
        }
    }
}

__device__ void _three_body(CUDA_FS_bond_list *bonds, c_number4 &F, c_number4 &T, c_number4 *forces, c_number4 *torques) {
    for(int pi = 0; pi < CUDAAllostericPatchySwapInteraction::MAX_PATCHES; pi++) {
        CUDA_FS_bond_list &bond_list = bonds[pi];

        for(int bi = 0; bi < bond_list.n_bonds; bi++) {
            CUDA_FS_bond &b1 = bond_list.bonds[bi];
            c_number curr_energy = b1.force.w;

            for(int bj = bi + 1; bj < bond_list.n_bonds; bj++) {
                CUDA_FS_bond &b2 = bond_list.bonds[bj];
                c_number other_energy = b2.force.w;

                // the factor 2 takes into account the fact that the total pair energy is always counted twice
                F.w += 2.f * MD_lambda[0] * curr_energy * other_energy;

                // b1
                c_number factor = -MD_lambda[0] * other_energy;

                c_number4 tmp_force = b1.force * factor;
                tmp_force.w = 0.f;

                F -= tmp_force;
                LR_atomicAddXYZ(forces + b1.q, tmp_force);

                T -= factor * b1.p_torque;
                LR_atomicAddXYZ(torques + b1.q, b1.q_torque_ref_frame * factor);

                // b2
                factor = -MD_lambda[0] * curr_energy;

                tmp_force = b2.force * factor;
                tmp_force.w = 0.f;

                F -= tmp_force;
                LR_atomicAddXYZ(forces + b2.q, tmp_force);

                T -= factor * b2.p_torque;
                LR_atomicAddXYZ(torques + b2.q, b2.q_torque_ref_frame * factor);
            }
        }
    }
}

/** @deprecated use the version with Verlet lists
 * computes the forces for a single particle with respect to all other
 * particles in the simulation. forces + second step without lists
 * @param poss positions of all particles in the simulation
 * @param orientations orientations of all particles in the simulation
 * @param forces net forces on all particles in the simulation
 * @param three_body_forces
 * @param torques
 * @param three_body_torques
 * @param box
 * @param patch_activations
 * @param particle_states
 */
__global__ void DPS_forces(c_number4 *poss,
                           GPU_quat *orientations,
                           c_number4 *forces,
                           c_number4 *three_body_forces,
                           c_number4 *torques,
                           c_number4 *three_body_torques,
                           CUDABox *box,
                           bool* patch_activations,
                           unsigned int *particle_states
) {
    if(IND >= MD_N[0]) return;

    c_number4 F = forces[IND];
    c_number4 T = torques[IND];
    c_number4 ppos = poss[IND];
    GPU_quat po = orientations[IND];
    c_number4 a1, a2, a3, b1, b2, b3;
    get_vectors_from_quat(po, a1, a2, a3);


    // create a list of all the bonds in this iteration
    CUDA_FS_bond_list bonds[CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

    // loop through every other particle in the simulation
    for(int j = 0; j < MD_N[0]; j++) {
        if(j != IND) {
            c_number4 qpos = poss[j];

            GPU_quat qo = orientations[j];
            get_vectors_from_quat(qo, b1, b2, b3);

            if(MD_is_KF[0]) {
                _patchy_KF_two_body_interaction(ppos,
                                                qpos,
                                                a1,
                                                a2,
                                                a3,
                                                b1,
                                                b2,
                                                b3,
                                                F,
                                                T,
                                                bonds,
                                                j,
                                                box,
                                                &patch_activations[IND],
                                                &patch_activations[j],
                                                particle_states[IND]);
            }
            else {
                _patchy_point_two_body_interaction(ppos,
                                                   qpos,
                                                   a1,
                                                   a2,
                                                   a3,
                                                   b1,
                                                   b2,
                                                   b3,
                                                   F,
                                                   T,
                                                   bonds,
                                                   j,
                                                   box,
                                                   &patch_activations[IND],
                                                   &patch_activations[j],
                                                   particle_states[IND]);
            }
        }
    }

    _three_body(bonds, F, T, three_body_forces, three_body_torques);

    forces[IND] = F;
    torques[IND] = _vectors_transpose_c_number4_product(a1, a2, a3, T);

    // apply new particle state to activations
    // there must be a better way to do this
    int p_type = get_particle_btype(ppos);
    for (int i = 0; i < MD_N_patches[p_type]; i++){
        patch_activations[IND + i] = MD_allosteric_controls[p_type][particle_states[IND]][i];
    }
//    memcpy(MD_allosteric_controls[p_type][particle_states[IND]],
//           patch_activations[IND], sizeof(bool) * )
}

/** forces + second step with verlet lists
 * Computes the forces on particle IND
 *
 * @param poss an array of c_number4 representing the positions of all particles. index with poss[IND]
 * @param orientations an array of quaternions representing the orientations of all particles. index with orientations[IND]
 * @param forces an array of forces
 * @param three_body_forces
 * @param torques
 * @param three_body_torques
 * @param matrix_neighs
 * @param c_number_neighs
 * @param box
 * @param p_activations
 * @param q_activations
 * @param p_state
 * @param q_state
 */
__global__ void DPS_forces(c_number4 *poss,
                           GPU_quat *orientations,
                           c_number4 *forces,
                           c_number4 *three_body_forces,
                           c_number4 *torques,
                           c_number4 *three_body_torques,
                           int *matrix_neighs,
                           int *c_number_neighs,
                           CUDABox *box,
                           bool* patch_activations,
                           unsigned int *particle_states) {
    if(IND >= MD_N[0]) return;

    c_number4 F = forces[IND]; // copy forces value to new variable
    c_number4 T = torques[IND]; // copy torques value to new variable
    c_number4 ppos = poss[IND]; // copy positions value to new variable
    GPU_quat po = orientations[IND];
    c_number4 a1, a2, a3, b1, b2, b3;
    get_vectors_from_quat(po, a1, a2, a3);

    // create a list of bonds
    CUDA_FS_bond_list bonds[CUDAAllostericPatchySwapInteraction::MAX_PATCHES];

    int num_neighs = c_number_neighs[IND];
    for(int j = 0; j < num_neighs; j++) {
        int k_index = matrix_neighs[j * MD_N[0] + IND];

        c_number4 qpos = poss[k_index];

        GPU_quat qo = orientations[k_index];
        get_vectors_from_quat(qo, b1, b2, b3);

        if(MD_is_KF[0]) {
            _patchy_KF_two_body_interaction(ppos,
                                            qpos,
                                            a1,
                                            a2,
                                            a3,
                                            b1,
                                            b2,
                                            b3,
                                            F,
                                            T,
                                            bonds,
                                            k_index,
                                            box,
                                            &patch_activations[IND], // pass memory address
                                            &patch_activations[k_index], // pass memory address
                                            particle_states[IND]);
        }
        else {
            _patchy_point_two_body_interaction(ppos,
                                               qpos,
                                               a1,
                                               a2,
                                               a3,
                                               b1,
                                               b2,
                                               b3,
                                               F,
                                               T,
                                               bonds,
                                               k_index,
                                               box,
                                               &patch_activations[IND], // pass memory address
                                               &patch_activations[k_index], // pass memory address
                                               particle_states[IND]);
        }
    }

    _three_body(bonds, F, T, three_body_forces, three_body_torques);

    forces[IND] = F;
    torques[IND] = _vectors_transpose_c_number4_product(a1, a2, a3, T);

    // apply new particle state to activations
    // there must be a better way to do this
    int p_type = get_particle_btype(ppos);
    if (particle_states[IND] != 0){
        printf("Particle id %i state %i", IND, particle_states[IND]);
    }
    for (int i = 0; i < MD_N_patches[p_type]; i++){
        patch_activations[IND + i] = MD_allosteric_controls[p_type][particle_states[IND]][i];
    }
//    memcpy(MD_allosteric_controls[p_type][particle_states[IND]],
//           patch_activations[IND], sizeof(bool) * )
}

/* END CUDA PART */

#define HALF_ISQRT3 0.28867513459481292f

CUDAAllostericPatchySwapInteraction::CUDAAllostericPatchySwapInteraction() :
        CUDABaseInteraction(),
        AllostericPatchySwapInteraction() {
    _step = 0;
}

CUDAAllostericPatchySwapInteraction::~CUDAAllostericPatchySwapInteraction() {
    if(_d_three_body_forces != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_three_body_forces));
    }

    if(_d_three_body_torques != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_three_body_torques));
    }

    if(_d_patchy_eps != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_patchy_eps));
        hipUnbindTexture(tex_patchy_eps);
    }

    if(_d_base_patches != nullptr) {
        CUDA_SAFE_CALL(hipFree(_d_base_patches));
        hipUnbindTexture(tex_base_patches);
    }

    if (_particle_binding_states != nullptr) {
        CUDA_SAFE_CALL(hipFree(_particle_binding_states));
    }

    if (_particle_activations != nullptr) {
        CUDA_SAFE_CALL(hipFree(_particle_activations));
    }
}

void CUDAAllostericPatchySwapInteraction::get_settings(input_file &inp) {
    AllostericPatchySwapInteraction::get_settings(inp);

    int sort_every = 0;
    getInputInt(&inp, "CUDA_sort_every", &sort_every, 0);
}

/**
 * copies data from CPU to GPU
 */
void CUDAAllostericPatchySwapInteraction::sync_GPU() {
    unsigned short* binding_states = new unsigned short[cudaParticleMemoryCount()];
    bool* activation_states = new bool[cudaParticleMemoryCount() * MAX_PATCHES];

    // loop particles
    for(int i = 0; i < realNumParticles(); i++) {
        AllostericPatchyParticle *particle = static_cast<AllostericPatchyParticle *>(CONFIG_INFO->particles()[i]);
        short particleState = 0; // init empty binding state
        // loop patches
        for (int p = 0; p < particle->patches.size(); p++) {
            // the short value particleState is a binary representation of
            // the particle state where each bit is a boolean value
            // representing a patch binding state

            // if patch is bound, set the bit particleState[p] to true
            if (particle->patches[p].bound) {
                particleState = particleState ^ (1 << p);
            }

            // get activation state
            bool activationState = particle->patches[p].is_active();
            activation_states[i * MAX_PATCHES + p] = activationState;
        }
        binding_states[i] = particleState;
    }
    // copy memory to gpu
    // destination, source
    CUDA_SAFE_CALL(hipMemcpy(_particle_activations,
                              activation_states,
                              getActivationStateArrayLength(),
                              hipMemcpyHostToDevice));
    // copy memory to gpu
    // destination, source
    CUDA_SAFE_CALL(hipMemcpy(_particle_binding_states,
                              binding_states,
                              getBindingStateArrayLength(),
                              hipMemcpyHostToDevice));
}

/**
 * copies data from GPU to CPU
 */
void CUDAAllostericPatchySwapInteraction::sync_host() {
    unsigned short* binding_states = new unsigned short[cudaParticleMemoryCount()];
    bool* activation_states = new bool[cudaParticleMemoryCount() * MAX_PATCHES];
    // copy memory from gpu to cpu
    // destination, source
    CUDA_SAFE_CALL(hipMemcpy(binding_states,
                              _particle_binding_states,
                              getBindingStateArrayLength(),
                              hipMemcpyDeviceToHost));
    // copy memory from gpu to cpu
    // destination, source
    CUDA_SAFE_CALL(hipMemcpy(activation_states,
                              _particle_activations,
                              getActivationStateArrayLength(),
                              hipMemcpyDeviceToHost));

    // loop particles
    for (int i = 0; i < realNumParticles(); i++){
        AllostericPatchyParticle* particle = static_cast<AllostericPatchyParticle*>(CONFIG_INFO->particles()[i]);
        for (int p = 0; p < particle->n_patches(); p++){
            // simply index the activation state array to get activation state
            bool activationState = activation_states[i * MAX_PATCHES + p];
            // update patch activation state
            particle->patches[p].set_active(activationState);
            // retrieve binding state of this patch by getting the p-th bit of binding_states[i]
            bool newBindingState = GET_BIT(binding_states[i], p) == 1;
            // can safely set patch binding directly b/c the above line will fix
            // activation states
            particle->patches[p].bound = newBindingState;
        }
        // DEBUG
        bool* bindingState = new bool[particle->n_patches()];
        for (int p = 0; p < particle->n_patches(); p++) {
            bindingState[p] = particle->patches[p].bound;
        }
        for (int p = 0; p < particle->n_patches(); p++) {
            bool computed_activation = particle->patch_status(bindingState, p);
            if (computed_activation != particle->patches[p].is_active()) {
                throw oxDNAException("Activation state %b of particle %d, patch %d is inconsistant with allosteric control conditional %s."
                                     "Binding state: %i",
                                     computed_activation, i, p,
                                     particle->patches[p].get_allosteric_conditional().c_str(),
                                     binding_states[i]);
            }
        }

    }
//    for(int i = 0; i < AllostericPatchySwapInteraction::_N; i++) {
//        AllostericPatchyParticle* particle = static_cast<AllostericPatchyParticle*>(CONFIG_INFO->particles()[i]);
//        short particleState;
//        CUDA_SAFE_CALL(hipMemcpy(particle_binding_states + i,
//                                  &particleState,
//                                  sizeof(short),
//                                  hipMemcpyDeviceToHost));
//
//        // loop patchesactivationState
//        for (int p = 0; p < particle->patches.size(); p++){
//            bool activationState;
//            CUDA_SAFE_CALL(hipMemcpy(activation_states + (MAX_PATCHES * i + p),
//                                      &activationState, sizeof(bool),
//                                      hipMemcpyDeviceToHost));
//
//            // the short value particleState is a binary representation of
//            // the particle state where each bit is a boolean value
//            // representing a patch binding state
//            bool newBindingState = particleState << i >= 2 << 15;
//            // set binding state
//            particle->patches[p].bound = newBindingState;
//
//            // set patch activation status
//            particle->patches[p].set_active(activationState);
//        }
//    }

}

void CUDAAllostericPatchySwapInteraction::cuda_init(c_number box_side, int N) {
    CUDABaseInteraction::cuda_init(box_side, N);
    AllostericPatchySwapInteraction::init();

    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_forces, N * sizeof(c_number4)));
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_three_body_torques, N * sizeof(c_number4)));

    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N), &N, sizeof(int)));

    COPY_NUMBER_TO_FLOAT(MD_sqr_rcut, _sqr_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sqr_rep_rcut, _sqr_rep_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sqr_patch_rcut, _sqr_patch_rcut);
    COPY_NUMBER_TO_FLOAT(MD_sigma_ss, _sigma_ss);
    COPY_NUMBER_TO_FLOAT(MD_rcut_ss, _rcut_ss);
    COPY_NUMBER_TO_FLOAT(MD_lambda, _lambda);
    COPY_NUMBER_TO_FLOAT(MD_A_part, _A_part);
    COPY_NUMBER_TO_FLOAT(MD_B_part, _B_part);
    COPY_NUMBER_TO_FLOAT(MD_spherical_E_cut, _spherical_E_cut);
    COPY_NUMBER_TO_FLOAT(MD_spherical_attraction_strength, _spherical_attraction_strength);

    // KF stuff
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_is_KF), &_is_KF, sizeof(bool)));

    if(_is_KF) {
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_power), &_patch_power, sizeof(int)));
        COPY_NUMBER_TO_FLOAT(MD_patch_pow_delta, _patch_pow_delta);
        COPY_NUMBER_TO_FLOAT(MD_patch_pow_cosmax, _patch_pow_cosmax);
        COPY_NUMBER_TO_FLOAT(MD_patch_angular_cutoff, _patch_angular_cutoff);
    }

    int N_strands;
    std::vector<BaseParticle *> particles(N);
    AllostericPatchySwapInteraction::read_topology(&N_strands, particles);

    // init particle state vars
    // allocate memory for particle binding states
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_particle_binding_states,
                                           getBindingStateArrayLength() * sizeof(unsigned int)));
    // allocate memory for patch activation states
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_particle_activations,
                                           getActivationStateArrayLength() * sizeof(bool)));
    bool* unbound_state = new bool[MAX_PATCHES];
    unsigned int* particle_states_empty = new unsigned int[N];
    bool* all_activation_states = new bool[MAX_PATCHES * N];
    std::fill(unbound_state, unbound_state + MAX_PATCHES, false);
    std::fill(particle_states_empty, particle_states_empty + N, 0);
    for (int i = 0; i < N; i++){
        AllostericPatchyParticle* pp = dynamic_cast<AllostericPatchyParticle*>(particles[i]);
        for (int x = 0; x < pp->n_patches(); x++){
            all_activation_states[i * MAX_PATCHES + x] = pp->patch_status(unbound_state, x);
        }
    }
    CUDA_SAFE_CALL(hipMemcpy(_particle_binding_states, particle_states_empty,
                              getBindingStateArrayLength() * sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipMemcpy(_particle_activations, all_activation_states,
                              getActivationStateArrayLength() * sizeof(bool), hipMemcpyHostToDevice));

    delete[] unbound_state;
    delete[] particle_states_empty;
    delete[] all_activation_states;

    for(auto particle : particles) {
        delete particle;
    }

    int N_species = this->_base_particle_types.size();
    if(N_species > MAX_SPECIES) {
        throw oxDNAException("PatchySwapInteraction: cannot simulate more than %d species. You can increase this number in the PatchySwapInteraction.h file", MAX_SPECIES);
    }
    uint n_patches[N_species];
    for (int i = 0; i < N_species; i++){
        n_patches[i] = _base_particle_types[i].patches.size();
    }

    // the following quantities are initialised by read_topology and hence have to be copied over to the GPU after its call
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patch_types), &_N_patch_types, sizeof(int)));
    CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_N_patches), n_patches, sizeof(int) * N_species));

    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_patchy_eps, _patchy_eps.size() * sizeof(float)));
    std::vector<float> h_patchy_eps(_patchy_eps.begin(), _patchy_eps.end());
    CUDA_SAFE_CALL(hipMemcpy(_d_patchy_eps, h_patchy_eps.data(), _patchy_eps.size() * sizeof(float), hipMemcpyHostToDevice));
    // bind member variable _d_patchy_eps to tex_patchy_eps
    CUDA_SAFE_CALL(hipBindTexture(NULL, tex_patchy_eps, _d_patchy_eps, _patchy_eps.size() * sizeof(float)));

    int N_base_patches = MAX_PATCHES * N_species;
    CUDA_SAFE_CALL(GpuUtils::LR_cudaMalloc(&_d_base_patches, N_base_patches * sizeof(float4)));
    std::vector<float4> h_base_patches(N_base_patches, make_float4(0., 0., 0., 0.));
    for(uint ns = 0; ns < N_species; ns++) {
        AllostericPatchyParticle& particle_type = _base_particle_types[ns];
        for(uint np = 0; np < particle_type.n_patches(); np++) {
            AllostericPatch& patch = particle_type.patches[np];
            float4 bp_f4 = make_float4(patch.position().x, patch.position().y, patch.position().z, 0.);
            h_base_patches[ns * MAX_PATCHES + np] = bp_f4;
        }
    }

    CUDA_SAFE_CALL(hipMemcpy(_d_base_patches, h_base_patches.data(), N_base_patches * sizeof(float4), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL(hipBindTexture(NULL, tex_base_patches, _d_base_patches, N_base_patches * sizeof(float4)));

    for(int i = 0; i < N_species; i++) {
        int n_patches = _base_particle_types[i].patches.size();

        if(n_patches > MAX_PATCHES) {
            throw oxDNAException("CUDAAllostericPatchySwapInteraction: cannot simulate particles with more than %d patches. You can increase this number in the AllostericPatchySwapInteraction.h file", MAX_PATCHES);
        }

        int patch_types[MAX_PATCHES];
        for(int p = 0; p < n_patches; p++) {
            // the patchy_epsilon matrix indexes by ID, not color!
            patch_types[p] = _base_particle_types[i].patches[p].get_id();
        }

        float4 base_patches[MAX_PATCHES];
        float4 patch_a1s[MAX_PATCHES];
        float4 patch_a2s[MAX_PATCHES];
        // allocate memory for patch position
        for(int p = 0; p < n_patches; p++) {
            // patch position
            LR_vector patch_position = _base_particle_types[i].patches[p].position();
            base_patches[p] = make_c_number4(patch_position.x, patch_position.y, patch_position.z, 0);
            // patch orientation
            LR_vector a1 = _base_particle_types[i].patches[p].a1();
            LR_vector a2 = _base_particle_types[i].patches[i].a2();
            patch_a1s[p] = make_c_number4(a1.x, a1.y, a1.z, 0);
            patch_a2s[p] = make_c_number4(a2.x, a2.y, a2.z, 0);

            // time to deal with allostery!
            bool patches_allosteric_flips[MAX_STATES][MAX_PATCHES];

            bool state[MAX_PATCHES];
            for (unsigned int q = 0; q < MAX_STATES; q++){
                // each unique state can be expressed as an MAX_STATES-digit binary number where
                // each digit is a patch binding state

                // first decode state
                for (int x = 0; x < MAX_PATCHES; x++){
                    state[x] = GET_BIT(q, x);
                }

                // encode flip value for each patch x in relation to q
                for (int x = 0; x < MAX_PATCHES; x++) {
                    // have to make a new pointer every time b/c the state change deconstructor deallocates
                    // the array memory
                    bool* stateptr = new bool[MAX_PATCHES];
                    std::copy(state, state + MAX_PATCHES, stateptr);
                    // get the particle state change originating at `state` when patch `x` is flipped
                    ParticleStateChange state_change(stateptr, MAX_PATCHES, x);
                    bool outcome = _base_particle_types[i].get_state_change_result(state_change, p);
                    printf("State # %i, Change %i: %s => (%i flip: %b)", q, x, state_change.toString().c_str(), p, outcome);
                    // get the state change result, specifically the effect on patch p
                    patches_allosteric_flips[q][x] = outcome;
                }
            }
            // I'm like 79% sure these values are right
            int allo_mem_count = sizeof(bool) * MAX_STATES * MAX_PATCHES;
            int allo_mem_offset = (i * MAX_PATCHES + p) * allo_mem_offset;
            // copy computed flips to array
            CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_allosteric_controls), patches_allosteric_flips, allo_mem_count, allo_mem_offset));
        }

        // fourth argument is the offset
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_patch_types), patch_types, sizeof(int) * n_patches, i * sizeof(int) * MAX_PATCHES));
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patch_a1s), patch_a1s, sizeof(float4) * n_patches, i * sizeof(float4) * MAX_PATCHES));
        CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(MD_base_patch_a2s), patch_a2s, sizeof(float4) * n_patches, i * sizeof(float4) * MAX_PATCHES));
    }
}

/**
 *
 * @param lists list of particles
 * @param d_poss probably an array of particle positions?
 * @param d_orientations probably an array of particle orientations?
 * @param d_forces
 * @param d_torques
 * @param d_bonds
 * @param d_box
 */
void CUDAAllostericPatchySwapInteraction::compute_forces(CUDABaseList *lists, c_number4 *d_poss, GPU_quat *d_orientations, c_number4 *d_forces, c_number4 *d_torques, LR_bonds *d_bonds, CUDABox *d_box) {
    int N = cudaParticleMemoryCount(); // number of particlesa
    // construct data structures for three-body computations
    thrust::device_ptr < c_number4 > t_forces = thrust::device_pointer_cast(d_forces);
    thrust::device_ptr < c_number4 > t_torques = thrust::device_pointer_cast(d_torques);
    thrust::device_ptr < c_number4 > t_three_body_forces = thrust::device_pointer_cast(_d_three_body_forces);
    thrust::device_ptr < c_number4 > t_three_body_torques = thrust::device_pointer_cast(_d_three_body_torques);
    thrust::fill_n(t_three_body_forces, N, make_c_number4(0, 0, 0, 0));
    thrust::fill_n(t_three_body_torques, N, make_c_number4(0, 0, 0, 0));


    CUDASimpleVerletList *_v_lists = dynamic_cast<CUDASimpleVerletList *>(lists);
    if(_v_lists != NULL) {
        if(_v_lists->use_edge()) throw oxDNAException("CUDAAllostericPatchySwapInteraction: use_edge is unsupported");
        else {
            DPS_forces
            <<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
                    (d_poss,
                     d_orientations,
                     d_forces,
                     _d_three_body_forces,
                     d_torques,
                     _d_three_body_torques,
                     _v_lists->d_matrix_neighs,
                     _v_lists->d_number_neighs,
                     d_box,
                     this->_particle_activations,
                     this->_particle_binding_states);
            CUT_CHECK_ERROR("DPS_forces simple_lists error");
        }
    }
    // NOTE: non-verlet version is @deprecated
    else {
        CUDANoList *_no_lists = dynamic_cast<CUDANoList *>(lists);
        if(_no_lists != NULL) {
            DPS_forces
            <<<_launch_cfg.blocks, _launch_cfg.threads_per_block>>>
                    (d_poss,
                     d_orientations,
                     d_forces,
                     _d_three_body_forces,
                     d_torques,
                     _d_three_body_torques,
                     d_box,
                     this->_particle_activations,
                     this->_particle_binding_states);
            CUT_CHECK_ERROR("DPS_forces no_lists error");
        }
    }

    // add the three body contributions to the two-body forces and torques
    thrust::transform(t_forces, t_forces + N, t_three_body_forces, t_forces, thrust::plus<c_number4>());
    thrust::transform(t_torques, t_torques + N, t_three_body_torques, t_torques, thrust::plus<c_number4>());
}


number CUDAAllostericPatchySwapInteraction::pair_interaction_nonbonded(BaseParticle *p, BaseParticle *q, bool compute_r, bool update_forces) {
    if(compute_r) {
        _computed_r = _box->min_image(p->pos, q->pos);
    }

    number energy = _spherical_patchy_two_body(p, q, false, update_forces);

    if(_is_KF) {
        energy += _patchy_two_body_KF(p, q, false, update_forces);
    }
    else {
        energy += _patchy_two_body_point(p, q, false, update_forces);
    }

    return energy;
}

void CUDAAllostericPatchySwapInteraction::begin_energy_computation() {

}